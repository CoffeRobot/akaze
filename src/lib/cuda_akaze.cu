#include "hip/hip_runtime.h"
#include <opencv2/features2d/features2d.hpp>
#include "cuda_akaze.h"
#include "cudautils.h"

#define CONVROW_W 160
#define CONVCOL_W 32
#define CONVCOL_H 40
#define CONVCOL_S 8

#define SCHARR_W 32
#define SCHARR_H 16

#define NLDSTEP_W 32
#define NLDSTEP_H 13

#define ORIENT_S (13 * 16)
#define EXTRACT_S 64

__device__ __constant__ float d_Kernel[21];
__device__ unsigned int d_PointCounter[1];

template <int RADIUS>
__global__ void ConvRowGPU(float *d_Result, float *d_Data, int width, int pitch,
                           int height) {
  __shared__ float data[CONVROW_W + 2 * RADIUS];
  const int tx = threadIdx.x;
  const int minx = blockIdx.x * CONVROW_W;
  const int maxx = min(minx + CONVROW_W, width);
  const int yptr = blockIdx.y * pitch;
  const int loadPos = minx + tx - RADIUS;
  const int writePos = minx + tx;

  if (loadPos < 0)
    data[tx] = d_Data[yptr];
  else if (loadPos >= width)
    data[tx] = d_Data[yptr + width - 1];
  else
    data[tx] = d_Data[yptr + loadPos];
  __syncthreads();
  if (writePos < maxx && tx < CONVROW_W) {
    float sum = 0.0f;
    for (int i = 0; i <= (2 * RADIUS); i++) sum += data[tx + i] * d_Kernel[i];
    d_Result[yptr + writePos] = sum;
  }
}

///////////////////////////////////////////////////////////////////////////////
// Column convolution filter
///////////////////////////////////////////////////////////////////////////////
template <int RADIUS>
__global__ void ConvColGPU(float *d_Result, float *d_Data, int width, int pitch,
                           int height) {
  __shared__ float data[CONVCOL_W * (CONVCOL_H + 2 * RADIUS)];
  const int tx = threadIdx.x;
  const int ty = threadIdx.y;
  const int miny = blockIdx.y * CONVCOL_H;
  const int maxy = min(miny + CONVCOL_H, height) - 1;
  const int totStart = miny - RADIUS;
  const int totEnd = maxy + RADIUS;
  const int colStart = blockIdx.x * CONVCOL_W + tx;
  const int colEnd = colStart + (height - 1) * pitch;
  const int smemStep = CONVCOL_W * CONVCOL_S;
  const int gmemStep = pitch * CONVCOL_S;

  if (colStart < width) {
    int smemPos = ty * CONVCOL_W + tx;
    int gmemPos = colStart + (totStart + ty) * pitch;
    for (int y = totStart + ty; y <= totEnd; y += blockDim.y) {
      if (y < 0)
        data[smemPos] = d_Data[colStart];
      else if (y >= height)
        data[smemPos] = d_Data[colEnd];
      else
        data[smemPos] = d_Data[gmemPos];
      smemPos += smemStep;
      gmemPos += gmemStep;
    }
  }
  __syncthreads();
  if (colStart < width) {
    int smemPos = ty * CONVCOL_W + tx;
    int gmemPos = colStart + (miny + ty) * pitch;
    for (int y = miny + ty; y <= maxy; y += blockDim.y) {
      float sum = 0.0f;
      for (int i = 0; i <= 2 * RADIUS; i++)
        sum += data[smemPos + i * CONVCOL_W] * d_Kernel[i];
      d_Result[gmemPos] = sum;
      smemPos += smemStep;
      gmemPos += gmemStep;
    }
  }
}

template <int RADIUS>
double SeparableFilter(CudaImage &inimg, CudaImage &outimg, CudaImage &temp,
                       float *h_Kernel) {
  int width = inimg.width;
  int pitch = inimg.pitch;
  int height = inimg.height;
  float *d_DataA = inimg.d_data;
  float *d_DataB = outimg.d_data;
  float *d_Temp = temp.d_data;
  if (d_DataA == NULL || d_DataB == NULL || d_Temp == NULL) {
    printf("SeparableFilter: missing data\n");
    return 0.0;
  }
  // TimerGPU timer0(0);
  const unsigned int kernelSize = (2 * RADIUS + 1) * sizeof(float);
  safeCall(hipMemcpyToSymbolAsync(HIP_SYMBOL(d_Kernel), h_Kernel, kernelSize));

  dim3 blockGridRows(iDivUp(width, CONVROW_W), height);
  dim3 threadBlockRows(CONVROW_W + 2 * RADIUS);
  ConvRowGPU<RADIUS> << <blockGridRows, threadBlockRows>>>
      (d_Temp, d_DataA, width, pitch, height);
  // checkMsg("ConvRowGPU() execution failed\n");
  // safeCall(hipDeviceSynchronize());
  dim3 blockGridColumns(iDivUp(width, CONVCOL_W), iDivUp(height, CONVCOL_H));
  dim3 threadBlockColumns(CONVCOL_W, CONVCOL_S);
  ConvColGPU<RADIUS> << <blockGridColumns, threadBlockColumns>>>
      (d_DataB, d_Temp, width, pitch, height);
  // checkMsg("ConvColGPU() execution failed\n");
  // safeCall(hipDeviceSynchronize());

  double gpuTime = 0;  // timer0.read();
#ifdef VERBOSE
  printf("SeparableFilter time =        %.2f ms\n", gpuTime);
#endif
  return gpuTime;
}

template <int RADIUS>
double LowPass(CudaImage &inimg, CudaImage &outimg, CudaImage &temp,
               double var) {
  float kernel[2 * RADIUS + 1];
  float kernelSum = 0.0f;
  for (int j = -RADIUS; j <= RADIUS; j++) {
    kernel[j + RADIUS] = (float)expf(-(double)j * j / 2.0 / var);
    kernelSum += kernel[j + RADIUS];
  }
  for (int j = -RADIUS; j <= RADIUS; j++) kernel[j + RADIUS] /= kernelSum;
  return SeparableFilter<RADIUS>(inimg, outimg, temp, kernel);
}

double LowPass(CudaImage &inimg, CudaImage &outimg, CudaImage &temp, double var,
               int kernsize) {
  if (kernsize <= 5)
    return LowPass<2>(inimg, outimg, temp, var);
  else if (kernsize <= 7)
    return LowPass<3>(inimg, outimg, temp, var);
  else if (kernsize <= 9)
    return LowPass<4>(inimg, outimg, temp, var);
  else {
    if (kernsize > 11)
      std::cerr << "Kernels larger than 11 not implemented" << std::endl;
    return LowPass<5>(inimg, outimg, temp, var);
  }
}

__global__ void Scharr(float *imgd, float *lxd, float *lyd, int width,
                       int pitch, int height) {
#define BW (SCHARR_W + 2)
  __shared__ float buffer[BW * (SCHARR_H + 2)];
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int x = blockIdx.x * SCHARR_W + tx;
  int y = blockIdx.y * SCHARR_H + ty;
  int xp = (x == 0 ? 1 : (x > width ? width - 2 : x - 1));
  int yp = (y == 0 ? 1 : (y > height ? height - 2 : y - 1));
  buffer[ty * BW + tx] = imgd[yp * pitch + xp];
  __syncthreads();
  if (x < width && y < height && tx < SCHARR_W && ty < SCHARR_H) {
    float *b = buffer + (ty + 1) * BW + (tx + 1);
    float ul = b[-BW - 1];
    float ur = b[-BW + 1];
    float ll = b[+BW - 1];
    float lr = b[+BW + 1];
    lxd[y * pitch + x] = 3.0f * (lr - ll + ur - ul) + 10.0f * (b[+1] - b[-1]);
    lyd[y * pitch + x] = 3.0f * (lr + ll - ur - ul) + 10.0f * (b[BW] - b[-BW]);
  }
}

double Scharr(CudaImage &img, CudaImage &lx, CudaImage &ly) {
  // TimerGPU timer0(0);
  dim3 blocks(iDivUp(img.width, SCHARR_W), iDivUp(img.height, SCHARR_H));
  dim3 threads(SCHARR_W + 2, SCHARR_H + 2);
  Scharr << <blocks, threads>>>
      (img.d_data, lx.d_data, ly.d_data, img.width, img.pitch, img.height);
  // checkMsg("Scharr() execution failed\n");
  // safeCall(hipDeviceSynchronize());
  double gpuTime = 0;  // timer0.read();
#ifdef VERBOSE
  printf("Scharr time          =        %.2f ms\n", gpuTime);
#endif
  return gpuTime;
}

__global__ void Flow(float *imgd, float *flowd, int width, int pitch,
                     int height, DIFFUSIVITY_TYPE type, float invk) {
#define BW (SCHARR_W + 2)
  __shared__ float buffer[BW * (SCHARR_H + 2)];
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int x = blockIdx.x * SCHARR_W + tx;
  int y = blockIdx.y * SCHARR_H + ty;
  int xp = (x == 0 ? 1 : (x > width ? width - 2 : x - 1));
  int yp = (y == 0 ? 1 : (y > height ? height - 2 : y - 1));
  buffer[ty * BW + tx] = imgd[yp * pitch + xp];
  __syncthreads();
  if (x < width && y < height && tx < SCHARR_W && ty < SCHARR_H) {
    float *b = buffer + (ty + 1) * BW + (tx + 1);
    float ul = b[-BW - 1];
    float ur = b[-BW + 1];
    float ll = b[+BW - 1];
    float lr = b[+BW + 1];
    float lx = 3.0f * (lr - ll + ur - ul) + 10.0f * (b[+1] - b[-1]);
    float ly = 3.0f * (lr + ll - ur - ul) + 10.0f * (b[BW] - b[-BW]);
    float dif2 = invk * (lx * lx + ly * ly);
    if (type == PM_G1)
      flowd[y * pitch + x] = exp(-dif2);
    else if (type == PM_G2)
      flowd[y * pitch + x] = 1.0f / (1.0f + dif2);
    else if (type == WEICKERT)
      flowd[y * pitch + x] = 1.0f - exp(-3.315 / (dif2 * dif2 * dif2 * dif2));
    else
      flowd[y * pitch + x] = 1.0f / sqrt(1.0f + dif2);
  }
}

double Flow(CudaImage &img, CudaImage &flow, DIFFUSIVITY_TYPE type,
            float kcontrast) {
  // TimerGPU timer0(0);
  dim3 blocks(iDivUp(img.width, SCHARR_W), iDivUp(img.height, SCHARR_H));
  dim3 threads(SCHARR_W + 2, SCHARR_H + 2);
  Flow << <blocks, threads>>> (img.d_data, flow.d_data, img.width, img.pitch,
                               img.height, type,
                               1.0f / (kcontrast * kcontrast));
  //  //checkMsg("Flow() execution failed\n");
  //  //safeCall(hipDeviceSynchronize());
  double gpuTime = 0;  // timer0.read();
#ifdef VERBOSE
  printf("Flow time =                   %.2f ms\n", gpuTime);
#endif
  return gpuTime;
}

struct NLDStruct {
  float *imgd;
  float *flod;
  float *temd;
  int width;
  int pitch;
  int height;
  float stepsize;
};

__global__ void NLDStep(float *imgd, float *flod, float *temd, int width,
                        int pitch, int height, float stepsize) {
#undef BW
#define BW (NLDSTEP_W + 2)
  __shared__ float ibuff[BW * (NLDSTEP_H + 2)];
  __shared__ float fbuff[BW * (NLDSTEP_H + 2)];
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int x = blockIdx.x * NLDSTEP_W + tx;
  int y = blockIdx.y * NLDSTEP_H + ty;
  int xp = (x == 0 ? 0 : (x > width ? width - 1 : x - 1));
  int yp = (y == 0 ? 0 : (y > height ? height - 1 : y - 1));
  ibuff[ty * BW + tx] = imgd[yp * pitch + xp];
  fbuff[ty * BW + tx] = flod[yp * pitch + xp];
  __syncthreads();
  if (tx < NLDSTEP_W && ty < NLDSTEP_H && x < width && y < height) {
    float *ib = ibuff + (ty + 1) * BW + (tx + 1);
    float *fb = fbuff + (ty + 1) * BW + (tx + 1);
    float ib0 = ib[0];
    float fb0 = fb[0];
    float xpos = (fb0 + fb[+1]) * (ib[+1] - ib0);
    float xneg = (fb0 + fb[-1]) * (ib0 - ib[-1]);
    float ypos = (fb0 + fb[+BW]) * (ib[+BW] - ib0);
    float yneg = (fb0 + fb[-BW]) * (ib0 - ib[-BW]);
    temd[y * pitch + x] = stepsize * (xpos - xneg + ypos - yneg);
    // s.imgd[y*s.pitch + x] = s.imgd[y*s.pitch+x] + s.stepsize*(xpos-xneg +
    // ypos-yneg);//temd[y*pitch + x];
  }
}

__global__ void NLDStep(NLDStruct s) {
#undef BW
#define BW (NLDSTEP_W + 2)
  __shared__ float ibuff[BW * (NLDSTEP_H + 2)];
  __shared__ float fbuff[BW * (NLDSTEP_H + 2)];
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int x = blockIdx.x * NLDSTEP_W + tx;
  int y = blockIdx.y * NLDSTEP_H + ty;
  int xp = (x == 0 ? 0 : (x > s.width ? s.width - 1 : x - 1));
  int yp = (y == 0 ? 0 : (y > s.height ? s.height - 1 : y - 1));
  ibuff[ty * BW + tx] = s.imgd[yp * s.pitch + xp];
  fbuff[ty * BW + tx] = s.flod[yp * s.pitch + xp];
  __syncthreads();
  if (tx < NLDSTEP_W && ty < NLDSTEP_H && x < s.width && y < s.height) {
    float *ib = ibuff + (ty + 1) * BW + (tx + 1);
    float *fb = fbuff + (ty + 1) * BW + (tx + 1);
    float ib0 = ib[0];
    float fb0 = fb[0];
    float xpos = (fb0 + fb[+1]) * (ib[+1] - ib0);
    float xneg = (fb0 + fb[-1]) * (ib0 - ib[-1]);
    float ypos = (fb0 + fb[+BW]) * (ib[+BW] - ib0);
    float yneg = (fb0 + fb[-BW]) * (ib0 - ib[-BW]);
    s.imgd[y * s.pitch + x] =
        s.imgd[y * s.pitch + x] +
        s.stepsize * (xpos - xneg + ypos - yneg);  // temd[y*pitch + x];
  }
}

__global__ void NLDUpdate(float *imgd, float *temd, int width, int pitch,
                          int height) {
  int x = blockIdx.x * 32 + threadIdx.x;
  int y = blockIdx.y * 16 + threadIdx.y;
  if (x < width && y < height) {
    int p = y * pitch + x;
    imgd[p] = imgd[p] + temd[p];
  }
}

double NLDStep(CudaImage &img, CudaImage &flow, CudaImage &temp,
               float stepsize) {
  // TimerGPU timer0(0);
  dim3 blocks0(iDivUp(img.width, NLDSTEP_W), iDivUp(img.height, NLDSTEP_H));
  dim3 threads0(NLDSTEP_W + 2, NLDSTEP_H + 2);
  NLDStruct s;
  s.imgd = img.d_data;
  s.flod = flow.d_data;
  s.temd = temp.d_data;
  s.width = img.width;
  s.pitch = img.pitch;
  s.height = img.height;
  s.stepsize = 0.5 * stepsize;
  // NLDStep<<<blocks0, threads0>>>(img.d_data, flow.d_data, temp.d_data,
  // img.width, img.pitch, img.height, 0.5f*stepsize);
  NLDStep << <blocks0, threads0>>> (s);
  // checkMsg("NLDStep() execution failed\n");
  // safeCall(hipDeviceSynchronize());
  dim3 blocks1(iDivUp(img.width, 32), iDivUp(img.height, 16));
  dim3 threads1(32, 16);
  // NLDUpdate<<<blocks1, threads1>>>(img.d_data, temp.d_data, img.width,
  // img.pitch, img.height);
  // checkMsg("NLDUpdate() execution failed\n");
  // safeCall(hipDeviceSynchronize());
  double gpuTime = 0;  // timer0.read();
#ifdef VERBOSE
  printf("NLDStep time =                %.2f ms\n", gpuTime);
#endif
  return gpuTime;
}

__global__ void HalfSample(float *iimd, float *oimd, int iwidth, int iheight,
                           int ipitch, int owidth, int oheight, int opitch) {
  __shared__ float buffer[16 * 33];
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int x = blockIdx.x * 16 + tx;
  int y = blockIdx.y * 16 + ty;
  if (x >= owidth || y >= oheight) return;
  float *ptri = iimd + (2 * y) * ipitch + (2 * x);
  if (2 * owidth == iwidth) {
    buffer[ty * 32 + tx] = owidth * (ptri[0] + ptri[1]);
    ptri += ipitch;
    buffer[ty * 32 + tx + 16] = owidth * (ptri[0] + ptri[1]);
    if (ty == 15) {
      ptri += ipitch;
      buffer[tx + 32 * 16] = owidth * (ptri[0] + ptri[1]);
    }
  } else {
    float f0 = owidth - x;
    float f2 = 1 + x;
    buffer[ty * 32 + tx] = f0 * ptri[0] + owidth * ptri[1] + f2 * ptri[2];
    ptri += ipitch;
    buffer[ty * 32 + tx + 16] = f0 * ptri[0] + owidth * ptri[1] + f2 * ptri[2];
    if (ty == 15 && 2 * oheight != iheight) {
      ptri += ipitch;
      buffer[tx + 32 * 16] = f0 * ptri[0] + owidth * ptri[1] + f2 * ptri[1];
    }
  }
  __syncthreads();
  float *buff = buffer + 32 * ty + tx;
  if (2 * oheight == iheight)
    oimd[y * opitch + x] = oheight * (buff[0] + buff[16]) / (iwidth * iheight);
  else {
    float f0 = oheight - y;
    float f2 = 1 + y;
    oimd[y * opitch + x] = (f0 * buff[0] + oheight * buff[16] + f2 * buff[32]) /
                           (iwidth * iheight);
  }
}

__global__ void HalfSample2(float *iimd, float *oimd, int ipitch, int owidth,
                            int oheight, int opitch) {
  int x = blockIdx.x * 32 + threadIdx.x;
  int y = blockIdx.y * 16 + threadIdx.y;
  if (x >= owidth || y >= oheight) return;
  float *ptr = iimd + (2 * y) * ipitch + (2 * x);
  oimd[y * opitch + x] =
      0.25f * (ptr[0] + ptr[1] + ptr[ipitch + 0] + ptr[ipitch + 1]);
}

double HalfSample(CudaImage &inimg, CudaImage &outimg) {
  // TimerGPU timer0(0);
  if (inimg.width == 2 * outimg.width && inimg.height == 2 * outimg.height) {
    dim3 blocks(iDivUp(outimg.width, 32), iDivUp(outimg.height, 16));
    dim3 threads(32, 16);
    HalfSample2 << <blocks, threads>>> (inimg.d_data, outimg.d_data,
                                        inimg.pitch, outimg.width,
                                        outimg.height, outimg.pitch);
  } else {
    dim3 blocks(iDivUp(outimg.width, 16), iDivUp(outimg.height, 16));
    dim3 threads(16, 16);
    HalfSample << <blocks, threads>>> (inimg.d_data, outimg.d_data, inimg.width,
                                       inimg.height, inimg.pitch, outimg.width,
                                       outimg.height, outimg.pitch);
  }
  // checkMsg("HalfSample() execution failed\n");
  // safeCall(hipDeviceSynchronize());
  double gpuTime = 0;  // timer0.read();
#ifdef VERBOSE
  printf("HalfSample time =             %.2f ms\n", gpuTime);
#endif
  return gpuTime;
}

double Copy(CudaImage &inimg, CudaImage &outimg) {
  // TimerGPU timer0(0);
  double gpuTime = 0;  // timer0.read();
  safeCall(hipMemcpy2DAsync(outimg.d_data, sizeof(float) * outimg.pitch,
                             inimg.d_data, sizeof(float) * outimg.pitch,
                             sizeof(float) * inimg.width, inimg.height,
                             hipMemcpyDeviceToDevice));
#ifdef VERBOSE
  printf("Copy time =                   %.2f ms\n", gpuTime);
#endif
  return gpuTime;
}

float *AllocBuffers(int width, int height, int num, int omax, int maxpts,
                    std::vector<CudaImage> &buffers, cv::KeyPoint *&pts,
                    CudaImage *&ims) {
  buffers.resize(omax * num);
  int w = width;
  int h = height;
  int p = iAlignUp(w, 128);
  int size = 0;
  for (int i = 0; i < omax; i++) {
    for (int j = 0; j < num; j++) {
      CudaImage &buf = buffers[i * num + j];
      buf.width = w;
      buf.height = h;
      buf.pitch = p;
      buf.d_data = (float *)((long)size);
      size += h * p;
    }
    w /= 2;
    h /= 2;
    p = iAlignUp(w, 128);
  }
  int ptsstart = size;
  size += sizeof(cv::KeyPoint) * maxpts / sizeof(float);
  int imgstart = size;
  size += sizeof(CudaImage) * (num * omax + sizeof(float) - 1) / sizeof(float);
  float *memory = NULL;
  size_t pitch;
  safeCall(hipMallocPitch((void **)&memory, &pitch, (size_t)4096,
                           (size + 4095) / 4096 * sizeof(float)));
  for (int i = 0; i < omax * num; i++) {
    CudaImage &buf = buffers[i];
    buf.d_data = memory + (long)buf.d_data;
  }
  pts = (cv::KeyPoint *)(memory + ptsstart);
  ims = (CudaImage *)(memory + imgstart);
  return memory;
}

void FreeBuffers(float *buffers) { safeCall(hipFree(buffers)); }

__device__ unsigned int d_Maxval[1];
__device__ int d_Histogram[512];

#define CONTRAST_W 64
#define CONTRAST_H 7
#define HISTCONT_W 64
#define HISTCONT_H 8
#define HISTCONT_R 4

__global__ void MaxContrast(float *imgd, float *cond, int width, int pitch,
                            int height) {
#define WID (CONTRAST_W + 2)
  __shared__ float buffer[WID * (CONTRAST_H + 2)];
  __shared__ unsigned int maxval[32];
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  if (tx < 32 && !ty) maxval[tx] = 0.0f;
  __syncthreads();
  int x = blockIdx.x * CONTRAST_W + tx;
  int y = blockIdx.y * CONTRAST_H + ty;
  if (x >= width || y >= height) return;
  float *b = buffer + ty * WID + tx;
  b[0] = imgd[y * pitch + x];
  __syncthreads();
  if (tx < CONTRAST_W && ty < CONTRAST_H && x < width - 2 && y < height - 2) {
    float dx = 3.0f * (b[0] - b[2] + b[2 * WID] - b[2 * WID + 2]) +
               10.0f * (b[WID] - b[WID + 2]);
    float dy = 3.0f * (b[0] + b[2] - b[2 * WID] - b[2 * WID + 2]) +
               10.0f * (b[1] - b[2 * WID + 1]);
    float grad = sqrt(dx * dx + dy * dy);
    cond[(y + 1) * pitch + (x + 1)] = grad;
    unsigned int *gradi = (unsigned int *)&grad;
    atomicMax(maxval + (tx & 31), *gradi);
  }
  __syncthreads();
  if (tx < 32 && !ty) atomicMax(d_Maxval, maxval[tx]);
}

__global__ void HistContrast(float *cond, int width, int pitch, int height,
                             float imaxval, int nbins) {
  __shared__ int hist[512];
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int i = ty * HISTCONT_W + tx;
  if (i < nbins) hist[i] = 0;
  __syncthreads();
  int x = blockIdx.x * HISTCONT_W + tx;
  int y = blockIdx.y * HISTCONT_H * HISTCONT_R + ty;
  if (x > 0 && x < width - 1) {
    for (int i = 0; i < HISTCONT_R; i++) {
      if (y > 0 && y < height - 1) {
        int idx = min((int)(nbins * cond[y * pitch + x] * imaxval), nbins - 1);
        atomicAdd(hist + idx, 1);
      }
      y += HISTCONT_H;
    }
  }
  __syncthreads();
  if (i < nbins && hist[i] > 0) atomicAdd(d_Histogram + i, hist[i]);
}

double ContrastPercentile(CudaImage &img, CudaImage &temp, CudaImage &blur,
                          float perc, int nbins, float &contrast) {
  // TimerGPU timer0(0);
  LowPass(img, blur, temp, 1.0f, 5);

  float h_Maxval = 0.0f;
  safeCall(hipMemcpyToSymbolAsync(HIP_SYMBOL(d_Maxval), &h_Maxval, sizeof(float)));
  dim3 blocks1(iDivUp(img.width, CONTRAST_W), iDivUp(img.height, CONTRAST_H));
  dim3 threads1(CONTRAST_W + 2, CONTRAST_H + 2);
  MaxContrast << <blocks1, threads1>>>
      (blur.d_data, temp.d_data, blur.width, blur.pitch, blur.height);
  // checkMsg("MaxContrast() execution failed\n");
  // safeCall(hipDeviceSynchronize());
  safeCall(hipMemcpyFromSymbolAsync(&h_Maxval, HIP_SYMBOL(d_Maxval), sizeof(float)));

  if (nbins > 512) {
    printf(
        "Warning: Largest number of possible bins in ContrastPercentile() is "
        "512\n");
    nbins = 512;
  }
  int h_Histogram[512];
  memset(h_Histogram, 0, nbins * sizeof(int));
  safeCall(
      hipMemcpyToSymbolAsync(HIP_SYMBOL(d_Histogram), h_Histogram, nbins * sizeof(int)));
  dim3 blocks2(iDivUp(temp.width, HISTCONT_W),
               iDivUp(temp.height, HISTCONT_H * HISTCONT_R));
  dim3 threads2(HISTCONT_W, HISTCONT_H);
  HistContrast << <blocks2, threads2>>> (temp.d_data, temp.width, temp.pitch,
                                         temp.height, 1.0f / h_Maxval, nbins);
  safeCall(
      hipMemcpyFromSymbolAsync(h_Histogram, HIP_SYMBOL(d_Histogram), nbins * sizeof(int)));

  int npoints = (temp.width - 2) * (temp.height - 2);
  int nthreshold = (int)(npoints * perc);
  int k = 0, nelements = 0;
  for (k = 0; nelements < nthreshold && k < nbins; k++)
    nelements += h_Histogram[k];
  contrast = (nelements < nthreshold ? 0.03f : h_Maxval * ((float)k / nbins));

  double gpuTime = 0;  // timer0.read();
#ifdef VERBOSE
  printf("ContrastPercentile time =     %.2f ms\n", gpuTime);
#endif
  return gpuTime;
}

__global__ void Derivate(float *imd, float *lxd, float *lyd, int width,
                         int pitch, int height, int step, float fac1,
                         float fac2) {
  int x = blockIdx.x * 32 + threadIdx.x;
  int y = blockIdx.y * 16 + threadIdx.y;
  if (x >= width || y >= height) return;
  int xl = (x < step ? step - x : x - step);
  int xh = (x >= width - step ? 2 * width - x - step - 2 : x + step);
  int yl = (y < step ? step - y : y - step);
  int yh = (y >= height - step ? 2 * height - y - step - 2 : y + step);
  float ul = imd[yl * pitch + xl];
  float ur = imd[yl * pitch + xh];
  float ll = imd[yh * pitch + xl];
  float lr = imd[yh * pitch + xh];
  float cl = imd[y * pitch + xl];
  float cr = imd[y * pitch + xh];
  lxd[y * pitch + x] = fac1 * (ur + lr - ul - ll) + fac2 * (cr - cl);
  float uc = imd[yl * pitch + x];
  float lc = imd[yh * pitch + x];
  lyd[y * pitch + x] = fac1 * (lr + ll - ur - ul) + fac2 * (lc - uc);
}

__global__ void HessianDeterminant(float *lxd, float *lyd, float *detd,
                                   int width, int pitch, int height, int step,
                                   float fac1, float fac2) {
  int x = blockIdx.x * 32 + threadIdx.x;
  int y = blockIdx.y * 16 + threadIdx.y;
  if (x >= width || y >= height) return;
  int xl = (x < step ? step - x : x - step);
  int xh = (x >= width - step ? 2 * width - x - step - 2 : x + step);
  int yl = (y < step ? step - y : y - step);
  int yh = (y >= height - step ? 2 * height - y - step - 2 : y + step);
  float ul = lxd[yl * pitch + xl];
  float ur = lxd[yl * pitch + xh];
  float ll = lxd[yh * pitch + xl];
  float lr = lxd[yh * pitch + xh];
  float cl = lxd[y * pitch + xl];
  float cr = lxd[y * pitch + xh];
  float lxx = fac1 * (ur + lr - ul - ll) + fac2 * (cr - cl);
  float uc = lxd[yl * pitch + x];
  float lc = lxd[yh * pitch + x];
  float lyx = fac1 * (lr + ll - ur - ul) + fac2 * (lc - uc);
  ul = lyd[yl * pitch + xl];
  ur = lyd[yl * pitch + xh];
  ll = lyd[yh * pitch + xl];
  lr = lyd[yh * pitch + xh];
  uc = lyd[yl * pitch + x];
  lc = lyd[yh * pitch + x];
  float lyy = fac1 * (lr + ll - ur - ul) + fac2 * (lc - uc);
  detd[y * pitch + x] = lxx * lyy - lyx * lyx;
}

double HessianDeterminant(CudaImage &img, CudaImage &lx, CudaImage &ly,
                          int step) {
  // TimerGPU timer0(0);
  float w = 10.0 / 3.0;
  float fac1 = 1.0 / (2.0 * (w + 2.0));
  float fac2 = w * fac1;
  dim3 blocks(iDivUp(img.width, 32), iDivUp(img.height, 16));
  dim3 threads(32, 16);
  Derivate << <blocks, threads>>> (img.d_data, lx.d_data, ly.d_data, img.width,
                                   img.pitch, img.height, step, fac1, fac2);
  // checkMsg("Derivate() execution failed\n");
  // safeCall(hipDeviceSynchronize());
  HessianDeterminant << <blocks, threads>>> (lx.d_data, ly.d_data, img.d_data,
                                             img.width, img.pitch, img.height,
                                             step, fac1, fac2);
  // checkMsg("HessianDeterminant() execution failed\n");
  // safeCall(hipDeviceSynchronize());
  double gpuTime = 0;  // timer0.read();
#ifdef VERBOSE
  printf("HessianDeterminant time =     %.2f ms\n", gpuTime);
#endif
  return gpuTime;
}

__global__ void FindExtrema(float *imd, float *imp, float *imn, int maxx,
                            int pitch, int maxy, int border, float dthreshold,
                            int scale, int octave, float size,
                            cv::KeyPoint *pts, int maxpts) {
  int x = blockIdx.x * 32 + threadIdx.x;
  int y = blockIdx.y * 16 + threadIdx.y;
  if (x < border || x >= maxx || y < border || y >= maxy) return;
  int p = y * pitch + x;
  float v = imd[p];
  if (v > dthreshold && v > imd[p - pitch - 1] && v > imd[p + pitch + 1] &&
      v > imd[p - pitch + 1] && v > imd[p - pitch + 1] && v > imd[p - 1] &&
      v > imd[p + 1] && v > imd[p + pitch] && v > imd[p - pitch] &&
      v >= imn[p] && v >= imp[p]) {
    float dx = 0.5f * (imd[p + 1] - imd[p - 1]);
    float dy = 0.5f * (imd[p + pitch] - imd[p - pitch]);
    float dxx = imd[p + 1] + imd[p - 1] - 2.0f * v;
    float dyy = imd[p + pitch] + imd[p - pitch] - 2.0f * v;
    float dxy = 0.25f * (imd[p + pitch + 1] + imd[p - pitch - 1] -
                         imd[p + pitch - 1] - imd[p - pitch + 1]);
    float det = dxx * dyy - dxy * dxy;
    float idet = (det != 0.0f ? 1.0f / det : 0.0f);
    float dst0 = idet * (dxy * dy - dyy * dx);
    float dst1 = idet * (dxy * dx - dxx * dy);
    if (dst0 >= -1.0f && dst0 <= 1.0f && dst1 >= -1.0f && dst1 <= 1.0f) {
      unsigned int idx = atomicInc(d_PointCounter, 0x7fffffff);
      if (idx < maxpts) {
        cv::KeyPoint &point = pts[idx];
        point.response = v;
        point.size = 2.0f * size;
        point.octave = octave;
        point.class_id = scale;
        int ratio = (1 << octave);
        point.pt.x = ratio * (x + dst0);
        point.pt.y = ratio * (y + dst1);
        point.angle = 0.0f;
        // printf("XXX %d %d %.2f %.2f XXX\n", x, y, dst0, dst1);
      }
    }
  }
}

double FindExtrema(CudaImage &img, CudaImage &imgp, CudaImage &imgn,
                   float border, float dthreshold, int scale, int octave,
                   float size, cv::KeyPoint *pts, int maxpts) {
  // TimerGPU timer0(0);
  dim3 blocks(iDivUp(img.width, 32), iDivUp(img.height, 16));
  dim3 threads(32, 16);
  int b = (int)border;
  FindExtrema << <blocks, threads>>>
      (img.d_data, imgp.d_data, imgn.d_data, img.width - b, img.pitch,
       img.height - b, b, dthreshold, scale, octave, size, pts, maxpts);
  // checkMsg("FindExtrema() execution failed\n");
  // safeCall(hipDeviceSynchronize());
  double gpuTime = 0;  // timer0.read();
#ifdef VERBOSE
  printf("FindExtrema time =            %.2f ms\n", gpuTime);
#endif
  return gpuTime;
}

void ClearPoints() {
  int totPts = 0;
  safeCall(hipMemcpyToSymbolAsync(HIP_SYMBOL(d_PointCounter), &totPts, sizeof(int)));
}

int GetPoints(std::vector<cv::KeyPoint> &h_pts, cv::KeyPoint *d_pts) {
  int numPts = 0;
  safeCall(hipMemcpyFromSymbolAsync(&numPts, HIP_SYMBOL(d_PointCounter), sizeof(int)));
  h_pts.resize(numPts);
  safeCall(hipMemcpy((float *)&h_pts[0], d_pts, sizeof(cv::KeyPoint) * numPts,
                      hipMemcpyDeviceToHost));
  return numPts;
}

__global__ void ExtractDescriptors(cv::KeyPoint *d_pts, CudaImage *d_imgs,
                                   float *_vals, int size2, int size3,
                                   int size4) {
  __shared__ float acc_vals_im[29 * EXTRACT_S];
  __shared__ float acc_vals_dx[29 * EXTRACT_S];
  __shared__ float acc_vals_dy[29 * EXTRACT_S];

  int p = blockIdx.x;

  float *vals = &_vals[p * 3 * 29];

  float iratio = 1.0f / (1 << d_pts[p].octave);
  int scale = (int)(0.5f * d_pts[p].size * iratio + 0.5f);
  float xf = d_pts[p].pt.x * iratio;
  float yf = d_pts[p].pt.y * iratio;
  float ang = d_pts[p].angle;
  float co = cos(ang);
  float si = sin(ang);
  int tx = threadIdx.x;
  int lev = d_pts[p].class_id;
  float *imd = d_imgs[4 * lev + 0].d_data;
  float *dxd = d_imgs[4 * lev + 2].d_data;
  float *dyd = d_imgs[4 * lev + 3].d_data;
  int pitch = d_imgs[4 * lev + 0].pitch;
  int winsize = max(3 * size3, 4 * size4);

  __shared__ int norm2[1];
  __shared__ int norm3[1];
  __shared__ int norm4[1];

  norm2[0] = 0;
  norm3[0] = 0;
  norm4[0] = 0;

  for (int i = 0; i < 29; ++i) {
    acc_vals_im[i * EXTRACT_S + tx] = 0.f;
    acc_vals_dx[i * EXTRACT_S + tx] = 0.f;
    acc_vals_dy[i * EXTRACT_S + tx] = 0.f;
  }

  __syncthreads();

  for (int i = tx; i < winsize * winsize; i += EXTRACT_S) {
    int y = i / winsize;
    int x = i - winsize * y;
    int m = max(x, y);
    if (m >= winsize) continue;
    int l = x - size2;
    int k = y - size2;
    int xp = (int)(xf + scale * (k * co - l * si) + 0.5f);
    int yp = (int)(yf + scale * (k * si + l * co) + 0.5f);
    int pos = yp * pitch + xp;
    float im = imd[pos];
    float dx = dxd[pos];
    float dy = dyd[pos];
    float rx = -dx * si + dy * co;
    float ry = dx * co + dy * si;

    if (m < 2 * size2) {
      int x2 = (x < size2 ? 0 : 1);
      int y2 = (y < size2 ? 0 : 1);
      //atomicAdd(norm2, (x < size2 && y < size2 ? 1 : 0));
      // Add 2x2
      acc_vals_im[(y2 * 2 + x2) + 29 * tx] += im;
      acc_vals_dx[(y2 * 2 + x2) + 29 * tx] += rx;
      acc_vals_dy[(y2 * 2 + x2) + 29 * tx] += ry;
    }
    if (m < 3 * size3) {
      int x3 = (x < size3 ? 0 : (x < 2 * size3 ? 1 : 2));
      int y3 = (y < size3 ? 0 : (y < 2 * size3 ? 1 : 2));
      //atomicAdd(norm3, (x < size3 && y < size3 ? 1 : 0));
      // Add 3x3
      acc_vals_im[(4 + y3 * 3 + x3) + 29 * tx] += im;
      acc_vals_dx[(4 + y3 * 3 + x3) + 29 * tx] += rx;
      acc_vals_dy[(4 + y3 * 3 + x3) + 29 * tx] += ry;
    }
    if (m < 4 * size4) {
      int x4 = (x < 2 * size4 ? (x < size4 ? 0 : 1) : (x < 3 * size4 ? 2 : 3));
      int y4 = (y < 2 * size4 ? (y < size4 ? 0 : 1) : (y < 3 * size4 ? 2 : 3));
      //atomicAdd(norm4, (x < size4 && y < size4 ? 1 : 0));
      // Add 4x4
      acc_vals_im[(4 + 9 + y4 * 4 + x4) + 29 * tx] += im;
      acc_vals_dx[(4 + 9 + y4 * 4 + x4) + 29 * tx] += rx;
      acc_vals_dy[(4 + 9 + y4 * 4 + x4) + 29 * tx] += ry;
    }
  }

  __syncthreads();

  // Reduce stuff
    for (int i = 0; i < 29; ++i) {
      if (tx < 32) {
        acc_vals_im[29 * tx + i] += acc_vals_im[29 * (tx + 32) + i];
        acc_vals_dx[29 * tx + i] += acc_vals_dx[29 * (tx + 32) + i];
        acc_vals_dy[29 * tx + i] += acc_vals_dy[29 * (tx + 32) + i];
      }
      if (tx < 16) {
        acc_vals_im[29 * tx + i] += acc_vals_im[29 * (tx + 16) + i];
        acc_vals_dx[29 * tx + i] += acc_vals_dx[29 * (tx + 16) + i];
        acc_vals_dy[29 * tx + i] += acc_vals_dy[29 * (tx + 16) + i];
      }
      if (tx < 8) {
        acc_vals_im[29 * tx + i] += acc_vals_im[29 * (tx + 8) + i];
        acc_vals_dx[29 * tx + i] += acc_vals_dx[29 * (tx + 8) + i];
        acc_vals_dy[29 * tx + i] += acc_vals_dy[29 * (tx + 8) + i];
      }
      if (tx < 4) {
        acc_vals_im[29 * tx + i] += acc_vals_im[29 * (tx + 4) + i];
        acc_vals_dx[29 * tx + i] += acc_vals_dx[29 * (tx + 4) + i];
        acc_vals_dy[29 * tx + i] += acc_vals_dy[29 * (tx + 4) + i];
      }
      if (tx < 2) {
        acc_vals_im[29 * tx + i] += acc_vals_im[29 * (tx + 2) + i];
        acc_vals_dx[29 * tx + i] += acc_vals_dx[29 * (tx + 2) + i];
        acc_vals_dy[29 * tx + i] += acc_vals_dy[29 * (tx + 2) + i];
      }
      if (tx < 1) {
        acc_vals_im[i] += acc_vals_im[29 + i];
        acc_vals_dx[i] += acc_vals_dx[29 + i];
        acc_vals_dy[i] += acc_vals_dy[29 + i];
      }
    }

  if (tx == 0) {
    for (int i = 0; i < 4; ++i) {
      vals[3 * i] = acc_vals_im[i];      // / (float)norm2[0];
      vals[3 * i + 1] = acc_vals_dx[i];  // / (float)norm2[0];
      vals[3 * i + 2] = acc_vals_dy[i];  // / (float)norm2[0];
    }
    for (int i = 0; i < 9; ++i) {
      vals[12 + 3 * i] = acc_vals_im[i + 4];      // / (float)norm3[0];
      vals[12 + 3 * i + 1] = acc_vals_dx[i + 4];  // / (float)norm3[0];
      vals[12 + 3 * i + 2] = acc_vals_dy[i + 4];  // / (float)norm3[0];
    }
    for (int i = 0; i < 16; ++i) {
      vals[39 + 3 * i] = acc_vals_im[i + 13];      // / (float)norm4[0];
      vals[39 + 3 * i + 1] = acc_vals_dx[i + 13];  // / (float)norm4[0];
      vals[39 + 3 * i + 2] = acc_vals_dy[i + 13];  // / (float)norm4[0];
    }
  }

  // acc_vals[0..28] is used to create feature vector
}

__global__ void ExtractDescriptorsSerial(cv::KeyPoint *d_pts, CudaImage *d_imgs,
                                         float *_vals, int size2, int size3,
                                         int size4) {
  __shared__ float acc_vals_im[29 * 1];
  __shared__ float acc_vals_dx[29 * 1];
  __shared__ float acc_vals_dy[29 * 1];

  int p = blockIdx.x;

  float *vals = &_vals[p * 3 * 29];

  float iratio = 1.0f / (1 << d_pts[p].octave);
  int scale = (int)(0.5f * d_pts[p].size * iratio + 0.5f);
  float xf = d_pts[p].pt.x * iratio;
  float yf = d_pts[p].pt.y * iratio;
  float ang = d_pts[p].angle;
  float co = cos(ang);
  float si = sin(ang);
  int tx = threadIdx.x;
  int lev = d_pts[p].class_id;
  float *imd = d_imgs[4 * lev + 0].d_data;
  float *dxd = d_imgs[4 * lev + 2].d_data;
  float *dyd = d_imgs[4 * lev + 3].d_data;
  int pitch = d_imgs[4 * lev + 0].pitch;
  int winsize = max(3 * size3, 4 * size4);

  for (int i = 0; i < 29; ++i) {
    acc_vals_im[i] = 0;
    acc_vals_dx[i] = 0;
    acc_vals_dy[i] = 0;
  }

  float norm2 = 0;
  float norm3 = 0;
  float norm4 = 0;

  for (int i = tx; i < winsize * winsize; i += 1) {
    int y = i / winsize;
    int x = i - winsize * y;
    int m = max(x, y);
    if (m >= winsize) continue;
    int l = x - size2;
    int k = y - size2;
    int xp = (int)(xf + scale * (k * co - l * si) + 0.5f);
    int yp = (int)(yf + scale * (k * si + l * co) + 0.5f);
    int pos = yp * pitch + xp;
    float im = imd[pos];
    float dx = dxd[pos];
    float dy = dyd[pos];
    float rx = -dx * si + dy * co;
    float ry = dx * co + dy * si;

    if (m < 2 * size2) {
      int x2 = (x < size2 ? 0 : 1);
      int y2 = (y < size2 ? 0 : 1);
      norm2 += (x < size2 && y < size2 ? 1 : 0);
      // Add 2x2
      acc_vals_im[y2 * 2 + x2] += im;
      acc_vals_dx[y2 * 2 + x2] += rx;
      acc_vals_dy[y2 * 2 + x2] += ry;
    }
    if (m < 3 * size3) {
      int x3 = (x < size3 ? 0 : (x < 2 * size3 ? 1 : 2));
      int y3 = (y < size3 ? 0 : (y < 2 * size3 ? 1 : 2));
      norm3 += (x < size3 && y < size3 ? 1 : 0);
      // Add 3x3
      acc_vals_im[4 + y3 * 3 + x3] += im;
      acc_vals_dx[4 + y3 * 3 + x3] += rx;
      acc_vals_dy[4 + y3 * 3 + x3] += ry;
    }
    if (m < 4 * size4) {
      int x4 = (x < 2 * size4 ? (x < size4 ? 0 : 1) : (x < 3 * size4 ? 2 : 3));
      int y4 = (y < 2 * size4 ? (y < size4 ? 0 : 1) : (y < 3 * size4 ? 2 : 3));
      norm4 += (x < size4 && y < size4 ? 1 : 0);
      // Add 4x4
      acc_vals_im[4 + 9 + y4 * 4 + x4] += im;
      acc_vals_dx[4 + 9 + y4 * 4 + x4] += rx;
      acc_vals_dy[4 + 9 + y4 * 4 + x4] += ry;
    }
  }

  __syncthreads();

  for (int i = 0; i < 4; ++i) {
    vals[3 * i] = acc_vals_im[i] / norm2;
    vals[3 * i + 1] = acc_vals_dx[i] / norm2;
    vals[3 * i + 2] = acc_vals_dy[i] / norm2;
  }
  for (int i = 0; i < 9; ++i) {
    vals[12 + 3 * i] = acc_vals_im[i + 4] / norm3;
    vals[12 + 3 * i + 1] = acc_vals_dx[i + 4] / norm3;
    vals[12 + 3 * i + 2] = acc_vals_dy[i + 4] / norm3;
  }
  for (int i = 0; i < 16; ++i) {
    vals[39 + 3 * i] = acc_vals_im[i + 13] / norm4;
    vals[39 + 3 * i + 1] = acc_vals_dx[i + 13] / norm4;
    vals[39 + 3 * i + 2] = acc_vals_dy[i + 13] / norm4;
  }
}

__global__ void BuildDescriptor(float *_valsim, unsigned char *_desc) {
  int p = blockIdx.x;

  float *valsim = &_valsim[3 * 29 * p];

  __shared__ unsigned char desc_s[64];

  unsigned char *desc = &_desc[61 * p];

  for (int i = 0; i < 64; ++i) {
    (desc_s)[i] = 0;
  }

  __syncthreads();

  // 2x2
  int cntr = 0;
  for (int j = 0; j < 4; ++j) {
    for (int i = j + 1; i < 4; ++i) {
      unsigned char im = valsim[3 * j] > valsim[3 * i] ? 1 : 0;
      desc_s[cntr >> 3] |= im << (cntr & 7);
      cntr++;
    }
  }
  for (int j = 0; j < 3; ++j) {
    for (int i = j + 1; i < 4; ++i) {
      unsigned char x = valsim[3 * j + 1] > valsim[3 * i + 1] ? 1 : 0;
      desc_s[cntr >> 3] |= x << (cntr & 7);
      cntr++;
    }
  }
  for (int j = 0; j < 3; ++j) {
    for (int i = j + 1; i < 4; ++i) {
      unsigned char y = valsim[3 * j + 2] > valsim[3 * i + 2] ? 1 : 0;
      desc_s[cntr >> 3] |= y << (cntr & 7);
      cntr++;
    }
  }

  // 3x3
  for (int j = 4; j < 12; ++j) {
    for (int i = j + 1; i < 13; ++i) {
      unsigned char im = valsim[3 * j] > valsim[3 * i] ? 1 : 0;
      desc_s[cntr >> 3] |= im << (cntr & 7);
      cntr++;
    }
  }
  for (int j = 4; j < 12; ++j) {
    for (int i = j + 1; i < 13; ++i) {
      unsigned char x = valsim[3 * j + 1] > valsim[3 * i + 1] ? 1 : 0;
      desc_s[cntr >> 3] |= x << (cntr & 7);
      cntr++;
    }
  }
  for (int j = 4; j < 12; ++j) {
    for (int i = j + 1; i < 13; ++i) {
      unsigned char y = valsim[3 * j + 2] > valsim[3 * i + 2] ? 1 : 0;
      desc_s[cntr >> 3] |= y << (cntr & 7);
      cntr++;
    }
  }

  // 4x4
  for (int j = 13; j < 28; ++j) {
    for (int i = j + 1; i < 29; ++i) {
      unsigned char im = valsim[3 * j] > valsim[3 * i] ? 1 : 0;
      desc_s[cntr >> 3] |= im << (cntr & 7);
      cntr++;
    }
  }
  for (int j = 13; j < 28; ++j) {
    for (int i = j + 1; i < 29; ++i) {
      unsigned char x = valsim[3 * j + 1] > valsim[3 * i + 1] ? 1 : 0;
      desc_s[cntr >> 3] |= x << (cntr & 7);
      cntr++;
    }
  }
  for (int j = 13; j < 28; ++j) {
    for (int i = j + 1; i < 29; ++i) {
      unsigned char y = valsim[3 * j + 2] > valsim[3 * i + 2] ? 1 : 0;
      desc_s[cntr >> 3] |= y << (cntr & 7);
      cntr++;
    }
  }

  __syncthreads();

  for (int i = 0; i < 61; ++i) {
    (desc)[i] = (desc_s)[i];
  }
}

double ExtractDescriptors(std::vector<cv::KeyPoint> &h_pts, cv::KeyPoint *d_pts,
                          std::vector<CudaImage> &h_imgs, CudaImage *d_imgs,
                          unsigned char *desc_h, int patsize) {
  int size2 = patsize;
  int size3 = (int)(2.0f * patsize / 3.0f + 0.5f);
  int size4 = (int)(0.5f * patsize + 0.5f);
  int numPts = h_pts.size();
  // TimerGPU timer0(0);
  dim3 blocks(numPts);
  dim3 threads(EXTRACT_S);

  float *vals_h = new float[3 * 29 * numPts];
  float *vals_d;
  hipMalloc(&vals_d, 3 * 29 * numPts * sizeof(float));

  ExtractDescriptors << <blocks, threads>>>
      (d_pts, d_imgs, vals_d, size2, size3, size4);
  // ExtractDescriptorsSerial << <blocks, 1>>>
  //    (d_pts, d_imgs, vals_d, size2, size3, size4);
  //hipMemcpy(vals_h, vals_d, 3 * 29 * numPts * sizeof(float),
  //           hipMemcpyDeviceToHost);

  int idx = 0;
  for (; idx < h_pts.size(); ++idx) {
    if ((int)h_pts[idx].pt.x == 840 && (int)h_pts[idx].pt.y == 45) break;
  }

  unsigned char *desc_d;
  hipMalloc(&desc_d, numPts * 61);
  hipMemsetAsync(desc_d, 0, numPts * 61);
  BuildDescriptor << <blocks, 1>>> (vals_d, desc_d);
  hipMemcpy(desc_h, desc_d, 61 * numPts, hipMemcpyDeviceToHost);

  float sum2 = 0, sum3 = 0, sum4 = 0;
  for (int i = 0; i < 4; ++i) {
    sum2 += vals_h[3 * 29 * idx + 3 * i];
  }
  for (int i = 0; i < 9; ++i) {
    sum3 += vals_h[3 * 29 * idx + 12 + 3 * i];
  }
  for (int i = 0; i < 16; ++i) {
    sum4 += vals_h[3 * 29 * idx + 39 + 3 * i];
  }

  std::cout << "sums: " << sum2 << " " << sum3 << " " << sum4 << std::endl;

  std::cout << "Keypoint idx: " << idx << std::endl;

  std::cout << "GPU output:\n";
  for (int i = 0; i < 3 * 29; ++i) {
    std::cout << vals_h[3 * 29 * idx + i] << " ";
  }
  std::cout << std::endl;
  for (int i = 0; i < 61; ++i) {
    std::cout << (unsigned int)desc_h[idx * 61 + i] << " ";
  }
  std::cout << "\n";

  ////checkMsg("ExtractDescriptors() execution failed\n");
  // safeCall(hipDeviceSynchronize());
  double gpuTime = 0;  // timer0.read();
#ifdef VERBOSE
  printf("ExtractDescriptors time =     %.2f ms\n", gpuTime);
#endif

  hipFree(vals_d);
  hipFree(desc_d);

  delete[] vals_h;

  return gpuTime;
}

__global__ void FindOrientation(cv::KeyPoint *d_pts, CudaImage *d_imgs) {
  __shared__ float resx[42], resy[42];
  __shared__ float re8x[42], re8y[42];
  int p = blockIdx.x;
  int tx = threadIdx.x;
  if (tx < 48) resx[tx] = resy[tx] = 0.0f;
  __syncthreads();
  int lev = d_pts[p].class_id;
  float *dxd = d_imgs[4 * lev + 2].d_data;
  float *dyd = d_imgs[4 * lev + 3].d_data;
  int pitch = d_imgs[4 * lev + 0].pitch;
  int octave = d_pts[p].octave;
  int step = (int)(0.5f * d_pts[p].size + 0.5f) >> octave;
  int x = (int)(d_pts[p].pt.x + 0.5f) >> octave;
  int y = (int)(d_pts[p].pt.y + 0.5f) >> octave;
  int i = (tx & 15) - 6;
  int j = (tx / 16) - 6;
  int r2 = i * i + j * j;
  if (r2 < 36) {
    float gweight = exp(-r2 / (2.5f * 2.5f * 2.0f));
    int pos = (y + step * j) * pitch + (x + step * i);
    float dx = gweight * dxd[pos];
    float dy = gweight * dyd[pos];
    float angle = atan2(dy, dx);
    int a = max(min((int)(angle * (21 / CV_PI)) + 21, 41), 0);
    atomicAdd(resx + a, dx);
    atomicAdd(resy + a, dy);
  }
  __syncthreads();
  if (tx < 42) {
    re8x[tx] = resx[tx];
    re8y[tx] = resy[tx];
    for (int k = tx + 1; k < tx + 7; k++) {
      re8x[tx] += resx[k < 42 ? k : k - 42];
      re8y[tx] += resy[k < 42 ? k : k - 42];
    }
  }
  __syncthreads();
  if (tx == 0) {
    float maxr = 0.0f;
    int maxk = 0;
    for (int k = 0; k < 42; k++) {
      float r = re8x[k] * re8x[k] + re8y[k] * re8y[k];
      if (r > maxr) {
        maxr = r;
        maxk = k;
      }
    }
    float angle = atan2(re8y[maxk], re8x[maxk]);
    d_pts[p].angle = (angle < 0.0f ? angle + 2.0f * CV_PI : angle);
    // printf("XXX %.2f %.2f %.2f\n", d_pts[p].pt.x, d_pts[p].pt.y,
    // d_pts[p].angle/CV_PI*180.0f);
  }
}

double FindOrientation(std::vector<cv::KeyPoint> &h_pts, cv::KeyPoint *d_pts,
                       std::vector<CudaImage> &h_imgs, CudaImage *d_imgs) {
  safeCall(hipMemcpyAsync(d_imgs, (float *)&h_imgs[0],
                           sizeof(CudaImage) * h_imgs.size(),
                           hipMemcpyHostToDevice));
  int numPts = h_pts.size();
  // TimerGPU timer0(0);
  dim3 blocks(numPts);
  dim3 threads(ORIENT_S);
  FindOrientation << <blocks, threads>>> (d_pts, d_imgs);
  // checkMsg("FindOrientation() execution failed\n");
  // safeCall(hipDeviceSynchronize());
  double gpuTime = 0;  // timer0.read();
#ifdef VERBOSE
  printf("FindOrientation time =        %.2f ms\n", gpuTime);
#endif
  return gpuTime;
}
