#include "hip/hip_runtime.h"
#include <opencv2/features2d/features2d.hpp>
#include "cuda_akaze.h"
#include "cudautils.h"

#define CONVROW_W 160
#define CONVCOL_W 32
#define CONVCOL_H 40
#define CONVCOL_S 8

#define SCHARR_W 32
#define SCHARR_H 16

#define NLDSTEP_W 32
#define NLDSTEP_H 13

#define ORIENT_S (13 * 16)
#define EXTRACT_S 64

static const float atan2_p1 = 0.9997878412794807f * (float)(180 / CV_PI);
static const float atan2_p3 = -0.3258083974640975f * (float)(180 / CV_PI);
static const float atan2_p5 = 0.1555786518463281f * (float)(180 / CV_PI);
static const float atan2_p7 = -0.04432655554792128f * (float)(180 / CV_PI);

__device__ __constant__ float d_Kernel[21];
__device__ unsigned int d_PointCounter[1];

__device__ __constant__ int comp_idx_1[61 * 8];
__device__ __constant__ int comp_idx_2[61 * 8];

__device__ __constant__ float norm_factors[29];

__device__ float fastAtan2(float y, float x) {
  float ax = std::abs(x), ay = std::abs(y);
  float a, c, c2;
  if (ax >= ay) {
    c = ay / (ax + (float)DBL_EPSILON);
    c2 = c * c;
    a = (((atan2_p7 * c2 + atan2_p5) * c2 + atan2_p3) * c2 + atan2_p1) * c;
  } else {
    c = ax / (ay + (float)DBL_EPSILON);
    c2 = c * c;
    a = 90.f -
        (((atan2_p7 * c2 + atan2_p5) * c2 + atan2_p3) * c2 + atan2_p1) * c;
  }
  if (x < 0) a = 180.f - a;
  if (y < 0) a = 360.f - a;
  return a;
}

template <int RADIUS>
__global__ void ConvRowGPU(float *d_Result, float *d_Data, int width, int pitch,
                           int height) {
  __shared__ float data[CONVROW_W + 2 * RADIUS];
  const int tx = threadIdx.x;
  const int minx = blockIdx.x * CONVROW_W;
  const int maxx = min(minx + CONVROW_W, width);
  const int yptr = blockIdx.y * pitch;
  const int loadPos = minx + tx - RADIUS;
  const int writePos = minx + tx;

  if (loadPos < 0)
    data[tx] = d_Data[yptr];
  else if (loadPos >= width)
    data[tx] = d_Data[yptr + width - 1];
  else
    data[tx] = d_Data[yptr + loadPos];
  __syncthreads();
  if (writePos < maxx && tx < CONVROW_W) {
    float sum = 0.0f;
    for (int i = 0; i <= (2 * RADIUS); i++) sum += data[tx + i] * d_Kernel[i];
    d_Result[yptr + writePos] = sum;
  }
}

///////////////////////////////////////////////////////////////////////////////
// Column convolution filter
///////////////////////////////////////////////////////////////////////////////
template <int RADIUS>
__global__ void ConvColGPU(float *d_Result, float *d_Data, int width, int pitch,
                           int height) {
  __shared__ float data[CONVCOL_W * (CONVCOL_H + 2 * RADIUS)];
  const int tx = threadIdx.x;
  const int ty = threadIdx.y;
  const int miny = blockIdx.y * CONVCOL_H;
  const int maxy = min(miny + CONVCOL_H, height) - 1;
  const int totStart = miny - RADIUS;
  const int totEnd = maxy + RADIUS;
  const int colStart = blockIdx.x * CONVCOL_W + tx;
  const int colEnd = colStart + (height - 1) * pitch;
  const int smemStep = CONVCOL_W * CONVCOL_S;
  const int gmemStep = pitch * CONVCOL_S;

  if (colStart < width) {
    int smemPos = ty * CONVCOL_W + tx;
    int gmemPos = colStart + (totStart + ty) * pitch;
    for (int y = totStart + ty; y <= totEnd; y += blockDim.y) {
      if (y < 0)
        data[smemPos] = d_Data[colStart];
      else if (y >= height)
        data[smemPos] = d_Data[colEnd];
      else
        data[smemPos] = d_Data[gmemPos];
      smemPos += smemStep;
      gmemPos += gmemStep;
    }
  }
  __syncthreads();
  if (colStart < width) {
    int smemPos = ty * CONVCOL_W + tx;
    int gmemPos = colStart + (miny + ty) * pitch;
    for (int y = miny + ty; y <= maxy; y += blockDim.y) {
      float sum = 0.0f;
      for (int i = 0; i <= 2 * RADIUS; i++)
        sum += data[smemPos + i * CONVCOL_W] * d_Kernel[i];
      d_Result[gmemPos] = sum;
      smemPos += smemStep;
      gmemPos += gmemStep;
    }
  }
}

template <int RADIUS>
double SeparableFilter(CudaImage &inimg, CudaImage &outimg, CudaImage &temp,
                       float *h_Kernel) {
  int width = inimg.width;
  int pitch = inimg.pitch;
  int height = inimg.height;
  float *d_DataA = inimg.d_data;
  float *d_DataB = outimg.d_data;
  float *d_Temp = temp.d_data;
  if (d_DataA == NULL || d_DataB == NULL || d_Temp == NULL) {
    printf("SeparableFilter: missing data\n");
    return 0.0;
  }
  // TimerGPU timer0(0);
  const unsigned int kernelSize = (2 * RADIUS + 1) * sizeof(float);
  safeCall(hipMemcpyToSymbolAsync(HIP_SYMBOL(d_Kernel), h_Kernel, kernelSize));

  dim3 blockGridRows(iDivUp(width, CONVROW_W), height);
  dim3 threadBlockRows(CONVROW_W + 2 * RADIUS);
  ConvRowGPU<RADIUS> << <blockGridRows, threadBlockRows>>>
      (d_Temp, d_DataA, width, pitch, height);
  // checkMsg("ConvRowGPU() execution failed\n");
  // safeCall(hipDeviceSynchronize());
  dim3 blockGridColumns(iDivUp(width, CONVCOL_W), iDivUp(height, CONVCOL_H));
  dim3 threadBlockColumns(CONVCOL_W, CONVCOL_S);
  ConvColGPU<RADIUS> << <blockGridColumns, threadBlockColumns>>>
      (d_DataB, d_Temp, width, pitch, height);
  // checkMsg("ConvColGPU() execution failed\n");
  // safeCall(hipDeviceSynchronize());

  double gpuTime = 0;  // timer0.read();
#ifdef VERBOSE
  printf("SeparableFilter time =        %.2f ms\n", gpuTime);
#endif
  return gpuTime;
}

template <int RADIUS>
double LowPass(CudaImage &inimg, CudaImage &outimg, CudaImage &temp,
               double var) {
  float kernel[2 * RADIUS + 1];
  float kernelSum = 0.0f;
  for (int j = -RADIUS; j <= RADIUS; j++) {
    kernel[j + RADIUS] = (float)expf(-(double)j * j / 2.0 / var);
    kernelSum += kernel[j + RADIUS];
  }
  for (int j = -RADIUS; j <= RADIUS; j++) kernel[j + RADIUS] /= kernelSum;
  return SeparableFilter<RADIUS>(inimg, outimg, temp, kernel);
}

double LowPass(CudaImage &inimg, CudaImage &outimg, CudaImage &temp, double var,
               int kernsize) {
  if (kernsize <= 5)
    return LowPass<2>(inimg, outimg, temp, var);
  else if (kernsize <= 7)
    return LowPass<3>(inimg, outimg, temp, var);
  else if (kernsize <= 9)
    return LowPass<4>(inimg, outimg, temp, var);
  else {
    if (kernsize > 11)
      std::cerr << "Kernels larger than 11 not implemented" << std::endl;
    return LowPass<5>(inimg, outimg, temp, var);
  }
}

__global__ void Scharr(float *imgd, float *lxd, float *lyd, int width,
                       int pitch, int height) {
#define BW (SCHARR_W + 2)
  __shared__ float buffer[BW * (SCHARR_H + 2)];
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int x = blockIdx.x * SCHARR_W + tx;
  int y = blockIdx.y * SCHARR_H + ty;
  int xp = (x == 0 ? 1 : (x > width ? width - 2 : x - 1));
  int yp = (y == 0 ? 1 : (y > height ? height - 2 : y - 1));
  buffer[ty * BW + tx] = imgd[yp * pitch + xp];
  __syncthreads();
  if (x < width && y < height && tx < SCHARR_W && ty < SCHARR_H) {
    float *b = buffer + (ty + 1) * BW + (tx + 1);
    float ul = b[-BW - 1];
    float ur = b[-BW + 1];
    float ll = b[+BW - 1];
    float lr = b[+BW + 1];
    lxd[y * pitch + x] = 3.0f * (lr - ll + ur - ul) + 10.0f * (b[+1] - b[-1]);
    lyd[y * pitch + x] = 3.0f * (lr + ll - ur - ul) + 10.0f * (b[BW] - b[-BW]);
  }
}

double Scharr(CudaImage &img, CudaImage &lx, CudaImage &ly) {
  // TimerGPU timer0(0);
  dim3 blocks(iDivUp(img.width, SCHARR_W), iDivUp(img.height, SCHARR_H));
  dim3 threads(SCHARR_W + 2, SCHARR_H + 2);
  Scharr << <blocks, threads>>>
      (img.d_data, lx.d_data, ly.d_data, img.width, img.pitch, img.height);
  // checkMsg("Scharr() execution failed\n");
  // safeCall(hipDeviceSynchronize());
  double gpuTime = 0;  // timer0.read();
#ifdef VERBOSE
  printf("Scharr time          =        %.2f ms\n", gpuTime);
#endif
  return gpuTime;
}

__global__ void Flow(float *imgd, float *flowd, int width, int pitch,
                     int height, DIFFUSIVITY_TYPE type, float invk) {
#define BW (SCHARR_W + 2)
  __shared__ float buffer[BW * (SCHARR_H + 2)];
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int x = blockIdx.x * SCHARR_W + tx;
  int y = blockIdx.y * SCHARR_H + ty;
  int xp = (x == 0 ? 1 : (x > width ? width - 2 : x - 1));
  int yp = (y == 0 ? 1 : (y > height ? height - 2 : y - 1));
  buffer[ty * BW + tx] = imgd[yp * pitch + xp];
  __syncthreads();
  if (x < width && y < height && tx < SCHARR_W && ty < SCHARR_H) {
    float *b = buffer + (ty + 1) * BW + (tx + 1);
    float ul = b[-BW - 1];
    float ur = b[-BW + 1];
    float ll = b[+BW - 1];
    float lr = b[+BW + 1];
    float lx = 3.0f * (lr - ll + ur - ul) + 10.0f * (b[+1] - b[-1]);
    float ly = 3.0f * (lr + ll - ur - ul) + 10.0f * (b[BW] - b[-BW]);
    float dif2 = invk * (lx * lx + ly * ly);
    if (type == PM_G1)
      flowd[y * pitch + x] = exp(-dif2);
    else if (type == PM_G2)
      flowd[y * pitch + x] = 1.0f / (1.0f + dif2);
    else if (type == WEICKERT)
      flowd[y * pitch + x] = 1.0f - exp(-3.315 / (dif2 * dif2 * dif2 * dif2));
    else
      flowd[y * pitch + x] = 1.0f / sqrt(1.0f + dif2);
  }
}

double Flow(CudaImage &img, CudaImage &flow, DIFFUSIVITY_TYPE type,
            float kcontrast) {
  // TimerGPU timer0(0);
  dim3 blocks(iDivUp(img.width, SCHARR_W), iDivUp(img.height, SCHARR_H));
  dim3 threads(SCHARR_W + 2, SCHARR_H + 2);
  Flow << <blocks, threads>>> (img.d_data, flow.d_data, img.width, img.pitch,
                               img.height, type,
                               1.0f / (kcontrast * kcontrast));
  //  //checkMsg("Flow() execution failed\n");
  //  //safeCall(hipDeviceSynchronize());
  double gpuTime = 0;  // timer0.read();
#ifdef VERBOSE
  printf("Flow time =                   %.2f ms\n", gpuTime);
#endif
  return gpuTime;
}

struct NLDStruct {
  float *imgd;
  float *flod;
  float *temd;
  int width;
  int pitch;
  int height;
  float stepsize;
};

__global__ void NLDStep(float *imgd, float *flod, float *temd, int width,
                        int pitch, int height, float stepsize) {
#undef BW
#define BW (NLDSTEP_W + 2)
  __shared__ float ibuff[BW * (NLDSTEP_H + 2)];
  __shared__ float fbuff[BW * (NLDSTEP_H + 2)];
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int x = blockIdx.x * NLDSTEP_W + tx;
  int y = blockIdx.y * NLDSTEP_H + ty;
  int xp = (x == 0 ? 0 : (x > width ? width - 1 : x - 1));
  int yp = (y == 0 ? 0 : (y > height ? height - 1 : y - 1));
  ibuff[ty * BW + tx] = imgd[yp * pitch + xp];
  fbuff[ty * BW + tx] = flod[yp * pitch + xp];
  __syncthreads();
  if (tx < NLDSTEP_W && ty < NLDSTEP_H && x < width && y < height) {
    float *ib = ibuff + (ty + 1) * BW + (tx + 1);
    float *fb = fbuff + (ty + 1) * BW + (tx + 1);
    float ib0 = ib[0];
    float fb0 = fb[0];
    float xpos = (fb0 + fb[+1]) * (ib[+1] - ib0);
    float xneg = (fb0 + fb[-1]) * (ib0 - ib[-1]);
    float ypos = (fb0 + fb[+BW]) * (ib[+BW] - ib0);
    float yneg = (fb0 + fb[-BW]) * (ib0 - ib[-BW]);
    temd[y * pitch + x] = stepsize * (xpos - xneg + ypos - yneg);
  }
}

__global__ void NLDStep(NLDStruct s) {
#undef BW
#define BW (NLDSTEP_W + 2)
  __shared__ float ibuff[BW * (NLDSTEP_H + 2)];
  __shared__ float fbuff[BW * (NLDSTEP_H + 2)];
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int x = blockIdx.x * NLDSTEP_W + tx;
  int y = blockIdx.y * NLDSTEP_H + ty;
  int xp = (x == 0 ? 0 : (x > s.width ? s.width - 1 : x - 1));
  int yp = (y == 0 ? 0 : (y > s.height ? s.height - 1 : y - 1));
  ibuff[ty * BW + tx] = s.imgd[yp * s.pitch + xp];
  fbuff[ty * BW + tx] = s.flod[yp * s.pitch + xp];
  __syncthreads();
  if (tx < NLDSTEP_W && ty < NLDSTEP_H && x < s.width && y < s.height) {
    float *ib = ibuff + (ty + 1) * BW + (tx + 1);
    float *fb = fbuff + (ty + 1) * BW + (tx + 1);
    float ib0 = ib[0];
    float fb0 = fb[0];
    float xpos = (fb0 + fb[+1]) * (ib[+1] - ib0);
    float xneg = (fb0 + fb[-1]) * (ib0 - ib[-1]);
    float ypos = (fb0 + fb[+BW]) * (ib[+BW] - ib0);
    float yneg = (fb0 + fb[-BW]) * (ib0 - ib[-BW]);
    s.imgd[y * s.pitch + x] =
        s.imgd[y * s.pitch + x] +
        s.stepsize * (xpos - xneg + ypos - yneg);  // temd[y*pitch + x];
  }
}

__global__ void NLDUpdate(float *imgd, float *temd, int width, int pitch,
                          int height) {
  int x = blockIdx.x * 32 + threadIdx.x;
  int y = blockIdx.y * 16 + threadIdx.y;
  if (x < width && y < height) {
    int p = y * pitch + x;
    imgd[p] = imgd[p] + temd[p];
  }
}

double NLDStep(CudaImage &img, CudaImage &flow, CudaImage &temp,
               float stepsize) {
  // TimerGPU timer0(0);
  dim3 blocks0(iDivUp(img.width, NLDSTEP_W), iDivUp(img.height, NLDSTEP_H));
  dim3 threads0(NLDSTEP_W + 2, NLDSTEP_H + 2);
  NLDStruct s;
  s.imgd = img.d_data;
  s.flod = flow.d_data;
  s.temd = temp.d_data;
  s.width = img.width;
  s.pitch = img.pitch;
  s.height = img.height;
  s.stepsize = 0.5 * stepsize;
  NLDStep<<<blocks0, threads0>>>(img.d_data, flow.d_data, temp.d_data,
                                 img.width, img.pitch, img.height, 0.5f*stepsize);
  // checkMsg("NLDStep() execution failed\n");
  // safeCall(hipDeviceSynchronize());
  dim3 blocks1(iDivUp(img.width, 32), iDivUp(img.height, 16));
  dim3 threads1(32, 16);
    NLDUpdate<<<blocks1, threads1>>>(img.d_data, temp.d_data, img.width,
                                     img.pitch, img.height);
  //  checkMsg("NLDUpdate() execution failed\n");
//    safeCall(hipDeviceSynchronize());
  double gpuTime = 0;  // timer0.read();
#ifdef VERBOSE
  printf("NLDStep time =                %.2f ms\n", gpuTime);
#endif
  return gpuTime;
}

__global__ void HalfSample(float *iimd, float *oimd, int iwidth, int iheight,
                           int ipitch, int owidth, int oheight, int opitch) {
  __shared__ float buffer[16 * 33];
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int x = blockIdx.x * 16 + tx;
  int y = blockIdx.y * 16 + ty;
  if (x >= owidth || y >= oheight) return;
  float *ptri = iimd + (2 * y) * ipitch + (2 * x);
    if (2*owidth==iwidth) {
        buffer[ty*32 + tx] = owidth*(ptri[0] + ptri[1]);
        ptri += ipitch;
        buffer[ty*32 + tx + 16] = owidth*(ptri[0] + ptri[1]);
        if (ty==15) {
            ptri += ipitch;
            buffer[tx + 32*16] = owidth*(ptri[0] + ptri[1]);
        } else if (y*2+3 == iheight) {
            ptri += ipitch;
            buffer[tx + 32*(ty+1)] = owidth*(ptri[0] + ptri[1]);
        }
    } else {
        float f0 = owidth - x;
        float f2 = 1 + x;
        buffer[ty*32 + tx] = f0*ptri[0] + owidth*ptri[1] + f2*ptri[2];
        ptri += ipitch;
        buffer[ty*32 + tx + 16] = f0*ptri[0] + owidth*ptri[1] + f2*ptri[2];
        if (ty==15 && 2*oheight!=iheight) {
            ptri += ipitch;
            buffer[tx + 32*16] = f0*ptri[0] + owidth*ptri[1] + f2*ptri[1];
        } else if (y*2+3 == iheight && 2*oheight!=iheight) {
            ptri += ipitch;
            buffer[tx + 32*(ty+1)] = f0*ptri[0] + owidth*ptri[1] + f2*ptri[2];
        }
    }
/*  if (2 * owidth == iwidth) {
    buffer[ty * 32 + tx] = owidth * (ptri[0] + ptri[1]);
    ptri += ipitch;
    buffer[ty * 32 + tx + 16] = owidth * (ptri[0] + ptri[1]);
    if (ty == 15) {
      ptri += ipitch;
      buffer[tx + 32 * 16] = owidth * (ptri[0] + ptri[1]);
    }
  } else {
    float f0 = owidth - x;
    float f2 = 1 + x;
    buffer[ty * 32 + tx] = f0 * ptri[0] + owidth * ptri[1] + f2 * ptri[2];
    ptri += ipitch;
    buffer[ty * 32 + tx + 16] = f0 * ptri[0] + owidth * ptri[1] + f2 * ptri[2];
    if (ty == 15 && 2 * oheight != iheight) {
      ptri += ipitch;
      buffer[tx + 32 * 16] = f0 * ptri[0] + owidth * ptri[1] + f2 * ptri[1];
    }
  }*/
  __syncthreads();
  float *buff = buffer + 32 * ty + tx;
  if (2 * oheight == iheight)
    oimd[y * opitch + x] = oheight * (buff[0] + buff[16]) / (iwidth * iheight);
  else {
    float f0 = oheight - y;
    float f2 = 1 + y;
    oimd[y * opitch + x] = (f0 * buff[0] + oheight * buff[16] + f2 * buff[32]) /
                           (iwidth * iheight);
  }
}

__global__ void HalfSample2(float *iimd, float *oimd, int ipitch, int owidth,
                            int oheight, int opitch) {
  int x = blockIdx.x * 32 + threadIdx.x;
  int y = blockIdx.y * 16 + threadIdx.y;
  if (x >= owidth || y >= oheight) return;
  float *ptr = iimd + (2 * y) * ipitch + (2 * x);
  oimd[y * opitch + x] =
      0.25f * (ptr[0] + ptr[1] + ptr[ipitch + 0] + ptr[ipitch + 1]);
}

double HalfSample(CudaImage &inimg, CudaImage &outimg) {
  // TimerGPU timer0(0);
  if (inimg.width == 2 * outimg.width && inimg.height == 2 * outimg.height) {
    dim3 blocks(iDivUp(outimg.width, 32), iDivUp(outimg.height, 16));
    dim3 threads(32, 16);
    HalfSample2 << <blocks, threads>>> (inimg.d_data, outimg.d_data,
                                        inimg.pitch, outimg.width,
                                        outimg.height, outimg.pitch);
  } else {
    dim3 blocks(iDivUp(outimg.width, 16), iDivUp(outimg.height, 16));
    dim3 threads(16, 16);
    HalfSample << <blocks, threads>>> (inimg.d_data, outimg.d_data, inimg.width,
                                       inimg.height, inimg.pitch, outimg.width,
                                       outimg.height, outimg.pitch);
  }
  // checkMsg("HalfSample() execution failed\n");
  // safeCall(hipDeviceSynchronize());
  double gpuTime = 0;  // timer0.read();
#ifdef VERBOSE
  printf("HalfSample time =             %.2f ms\n", gpuTime);
#endif
  return gpuTime;
}

double Copy(CudaImage &inimg, CudaImage &outimg) {
  // TimerGPU timer0(0);
  double gpuTime = 0;  // timer0.read();
  safeCall(hipMemcpy2DAsync(outimg.d_data, sizeof(float) * outimg.pitch,
                             inimg.d_data, sizeof(float) * outimg.pitch,
                             sizeof(float) * inimg.width, inimg.height,
                             hipMemcpyDeviceToDevice));
#ifdef VERBOSE
  printf("Copy time =                   %.2f ms\n", gpuTime);
#endif
  return gpuTime;
}

float *AllocBuffers(int width, int height, int num, int omax, int maxpts,
                    std::vector<CudaImage> &buffers, cv::KeyPoint *&pts,
                    CudaImage *&ims) {
  buffers.resize(omax * num);
  int w = width;
  int h = height;
  int p = iAlignUp(w, 128);
  int size = 0;
  for (int i = 0; i < omax; i++) {
    for (int j = 0; j < num; j++) {
      CudaImage &buf = buffers[i * num + j];
      buf.width = w;
      buf.height = h;
      buf.pitch = p;
      buf.d_data = (float *)((long)size);
      size += h * p;
    }
    w /= 2;
    h /= 2;
    p = iAlignUp(w, 128);
  }
  int ptsstart = size;
  size += sizeof(cv::KeyPoint) * maxpts / sizeof(float);
  int imgstart = size;
  size += sizeof(CudaImage) * (num * omax + sizeof(float) - 1) / sizeof(float);
  float *memory = NULL;
  size_t pitch;
  safeCall(hipMallocPitch((void **)&memory, &pitch, (size_t)4096,
                           (size + 4095) / 4096 * sizeof(float)));
  for (int i = 0; i < omax * num; i++) {
    CudaImage &buf = buffers[i];
    buf.d_data = memory + (long)buf.d_data;
  }
  pts = (cv::KeyPoint *)(memory + ptsstart);
  ims = (CudaImage *)(memory + imgstart);
  return memory;
}

void FreeBuffers(float *buffers) { safeCall(hipFree(buffers)); }

__device__ unsigned int d_Maxval[1];
__device__ int d_Histogram[512];

#define CONTRAST_W 64
#define CONTRAST_H 7
#define HISTCONT_W 64
#define HISTCONT_H 8
#define HISTCONT_R 4

__global__ void MaxContrast(float *imgd, float *cond, int width, int pitch,
                            int height) {
#define WID (CONTRAST_W + 2)
  __shared__ float buffer[WID * (CONTRAST_H + 2)];
  __shared__ unsigned int maxval[32];
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  if (tx < 32 && !ty) maxval[tx] = 0.0f;
  __syncthreads();
  int x = blockIdx.x * CONTRAST_W + tx;
  int y = blockIdx.y * CONTRAST_H + ty;
  if (x >= width || y >= height) return;
  float *b = buffer + ty * WID + tx;
  b[0] = imgd[y * pitch + x];
  __syncthreads();
  if (tx < CONTRAST_W && ty < CONTRAST_H && x < width - 2 && y < height - 2) {
    float dx = 3.0f * (b[0] - b[2] + b[2 * WID] - b[2 * WID + 2]) +
               10.0f * (b[WID] - b[WID + 2]);
    float dy = 3.0f * (b[0] + b[2] - b[2 * WID] - b[2 * WID + 2]) +
               10.0f * (b[1] - b[2 * WID + 1]);
    float grad = sqrt(dx * dx + dy * dy);
    cond[(y + 1) * pitch + (x + 1)] = grad;
    unsigned int *gradi = (unsigned int *)&grad;
    atomicMax(maxval + (tx & 31), *gradi);
  }
  __syncthreads();
  if (tx < 32 && !ty) atomicMax(d_Maxval, maxval[tx]);
}

__global__ void HistContrast(float *cond, int width, int pitch, int height,
                             float imaxval, int nbins) {
  __shared__ int hist[512];
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int i = ty * HISTCONT_W + tx;
  if (i < nbins) hist[i] = 0;
  __syncthreads();
  int x = blockIdx.x * HISTCONT_W + tx;
  int y = blockIdx.y * HISTCONT_H * HISTCONT_R + ty;
  if (x > 0 && x < width - 1) {
    for (int i = 0; i < HISTCONT_R; i++) {
      if (y > 0 && y < height - 1) {
        int idx = min((int)(nbins * cond[y * pitch + x] * imaxval), nbins - 1);
        atomicAdd(hist + idx, 1);
      }
      y += HISTCONT_H;
    }
  }
  __syncthreads();
  if (i < nbins && hist[i] > 0) atomicAdd(d_Histogram + i, hist[i]);
}

double ContrastPercentile(CudaImage &img, CudaImage &temp, CudaImage &blur,
                          float perc, int nbins, float &contrast) {
  // TimerGPU timer0(0);
  LowPass(img, blur, temp, 1.0f, 5);

  float h_Maxval = 0.0f;
  safeCall(hipMemcpyToSymbolAsync(HIP_SYMBOL(d_Maxval), &h_Maxval, sizeof(float)));
  dim3 blocks1(iDivUp(img.width, CONTRAST_W), iDivUp(img.height, CONTRAST_H));
  dim3 threads1(CONTRAST_W + 2, CONTRAST_H + 2);
  MaxContrast << <blocks1, threads1>>>
      (blur.d_data, temp.d_data, blur.width, blur.pitch, blur.height);
  // checkMsg("MaxContrast() execution failed\n");
  // safeCall(hipDeviceSynchronize());
  safeCall(hipMemcpyFromSymbolAsync(&h_Maxval, HIP_SYMBOL(d_Maxval), sizeof(float)));

  if (nbins > 512) {
    printf(
        "Warning: Largest number of possible bins in ContrastPercentile() is "
        "512\n");
    nbins = 512;
  }
  int h_Histogram[512];
  memset(h_Histogram, 0, nbins * sizeof(int));
  safeCall(
      hipMemcpyToSymbolAsync(HIP_SYMBOL(d_Histogram), h_Histogram, nbins * sizeof(int)));
  dim3 blocks2(iDivUp(temp.width, HISTCONT_W),
               iDivUp(temp.height, HISTCONT_H * HISTCONT_R));
  dim3 threads2(HISTCONT_W, HISTCONT_H);
  HistContrast << <blocks2, threads2>>> (temp.d_data, temp.width, temp.pitch,
                                         temp.height, 1.0f / h_Maxval, nbins);
  safeCall(
      hipMemcpyFromSymbolAsync(h_Histogram, HIP_SYMBOL(d_Histogram), nbins * sizeof(int)));

  int npoints = (temp.width - 2) * (temp.height - 2);
  int nthreshold = (int)(npoints * perc);
  int k = 0, nelements = 0;
  for (k = 0; nelements < nthreshold && k < nbins; k++)
    nelements += h_Histogram[k];
  contrast = (nelements < nthreshold ? 0.03f : h_Maxval * ((float)k / nbins));

  double gpuTime = 0;  // timer0.read();
#ifdef VERBOSE
  printf("ContrastPercentile time =     %.2f ms\n", gpuTime);
#endif
  return gpuTime;
}

__global__ void Derivate(float *imd, float *lxd, float *lyd, int width,
                         int pitch, int height, int step, float fac1,
                         float fac2) {
  int x = blockIdx.x * 32 + threadIdx.x;
  int y = blockIdx.y * 16 + threadIdx.y;
  if (x >= width || y >= height) return;
  int xl = (x < step ? step - x : x - step);
  int xh = (x >= width - step ? 2 * width - x - step - 2 : x + step);
  int yl = (y < step ? step - y : y - step);
  int yh = (y >= height - step ? 2 * height - y - step - 2 : y + step);
  float ul = imd[yl * pitch + xl];
  float ur = imd[yl * pitch + xh];
  float ll = imd[yh * pitch + xl];
  float lr = imd[yh * pitch + xh];
  float cl = imd[y * pitch + xl];
  float cr = imd[y * pitch + xh];
  lxd[y * pitch + x] = fac1 * (ur + lr - ul - ll) + fac2 * (cr - cl);
  float uc = imd[yl * pitch + x];
  float lc = imd[yh * pitch + x];
  lyd[y * pitch + x] = fac1 * (lr + ll - ur - ul) + fac2 * (lc - uc);
}

__global__ void HessianDeterminant(float *lxd, float *lyd, float *detd,
                                   int width, int pitch, int height, int step,
                                   float fac1, float fac2) {
  int x = blockIdx.x * 32 + threadIdx.x;
  int y = blockIdx.y * 16 + threadIdx.y;
  if (x >= width || y >= height) return;
  int xl = (x < step ? step - x : x - step);
  int xh = (x >= width - step ? 2 * width - x - step - 2 : x + step);
  int yl = (y < step ? step - y : y - step);
  int yh = (y >= height - step ? 2 * height - y - step - 2 : y + step);
  float ul = lxd[yl * pitch + xl];
  float ur = lxd[yl * pitch + xh];
  float ll = lxd[yh * pitch + xl];
  float lr = lxd[yh * pitch + xh];
  float cl = lxd[y * pitch + xl];
  float cr = lxd[y * pitch + xh];
  float lxx = fac1 * (ur + lr - ul - ll) + fac2 * (cr - cl);
  float uc = lxd[yl * pitch + x];
  float lc = lxd[yh * pitch + x];
  float lyx = fac1 * (lr + ll - ur - ul) + fac2 * (lc - uc);
  ul = lyd[yl * pitch + xl];
  ur = lyd[yl * pitch + xh];
  ll = lyd[yh * pitch + xl];
  lr = lyd[yh * pitch + xh];
  uc = lyd[yl * pitch + x];
  lc = lyd[yh * pitch + x];
  float lyy = fac1 * (lr + ll - ur - ul) + fac2 * (lc - uc);
  detd[y * pitch + x] = lxx * lyy - lyx * lyx;
}

double HessianDeterminant(CudaImage &img, CudaImage &lx, CudaImage &ly,
                          int step) {
  // TimerGPU timer0(0);
  float w = 10.0 / 3.0;
  float fac1 = 1.0 / (2.0 * (w + 2.0));
  float fac2 = w * fac1;
  dim3 blocks(iDivUp(img.width, 32), iDivUp(img.height, 16));
  dim3 threads(32, 16);
  Derivate << <blocks, threads>>> (img.d_data, lx.d_data, ly.d_data, img.width,
                                   img.pitch, img.height, step, fac1, fac2);
  // checkMsg("Derivate() execution failed\n");
  // safeCall(hipDeviceSynchronize());
  HessianDeterminant << <blocks, threads>>> (lx.d_data, ly.d_data, img.d_data,
                                             img.width, img.pitch, img.height,
                                             step, fac1, fac2);
  // checkMsg("HessianDeterminant() execution failed\n");
  // safeCall(hipDeviceSynchronize());
  double gpuTime = 0;  // timer0.read();
#ifdef VERBOSE
  printf("HessianDeterminant time =     %.2f ms\n", gpuTime);
#endif
  return gpuTime;
}

__global__ void FindExtrema(float *imd, float *imp, float *imn, int maxx,
                            int pitch, int maxy, int border, float dthreshold,
                            int scale, int octave, float size,
                            cv::KeyPoint *pts, int maxpts) {
  int x = blockIdx.x * 32 + threadIdx.x;
  int y = blockIdx.y * 16 + threadIdx.y;
  if (x < border || x >= maxx || y < border || y >= maxy) return;
  int p = y * pitch + x;
  float v = imd[p];
  if (v > dthreshold && v > imd[p - pitch - 1] && v > imd[p + pitch + 1] &&
      v > imd[p - pitch + 1] && v > imd[p - pitch + 1] && v > imd[p - 1] &&
      v > imd[p + 1] && v > imd[p + pitch] && v > imd[p - pitch] &&
      v >= imn[p] && v >= imp[p]) {
    float dx = 0.5f * (imd[p + 1] - imd[p - 1]);
    float dy = 0.5f * (imd[p + pitch] - imd[p - pitch]);
    float dxx = imd[p + 1] + imd[p - 1] - 2.0f * v;
    float dyy = imd[p + pitch] + imd[p - pitch] - 2.0f * v;
    float dxy = 0.25f * (imd[p + pitch + 1] + imd[p - pitch - 1] -
                         imd[p + pitch - 1] - imd[p - pitch + 1]);
    float det = dxx * dyy - dxy * dxy;
    float idet = (det != 0.0f ? 1.0f / det : 0.0f);
    float dst0 = idet * (dxy * dy - dyy * dx);
    float dst1 = idet * (dxy * dx - dxx * dy);
    if (dst0 >= -1.0f && dst0 <= 1.0f && dst1 >= -1.0f && dst1 <= 1.0f) {
      unsigned int idx = atomicInc(d_PointCounter, 0x7fffffff);
      if (idx < maxpts) {
        cv::KeyPoint &point = pts[idx];
        point.response = v;
        point.size = 2.0f * size;
        point.octave = octave;
        point.class_id = scale;
        int ratio = (1 << octave);
        point.pt.x = ratio * (x + dst0);
        point.pt.y = ratio * (y + dst1);
        point.angle = 0.0f;
        // printf("XXX %d %d %.2f %.2f XXX\n", x, y, dst0, dst1);
      }
    }
  }
}

double FindExtrema(CudaImage &img, CudaImage &imgp, CudaImage &imgn,
                   float border, float dthreshold, int scale, int octave,
                   float size, cv::KeyPoint *pts, int maxpts) {
  // TimerGPU timer0(0);
  dim3 blocks(iDivUp(img.width, 32), iDivUp(img.height, 16));
  dim3 threads(32, 16);
  int b = (int)border;
  FindExtrema << <blocks, threads>>>
      (img.d_data, imgp.d_data, imgn.d_data, img.width - b, img.pitch,
       img.height - b, b, dthreshold, scale, octave, size, pts, maxpts);
  // checkMsg("FindExtrema() execution failed\n");
  // safeCall(hipDeviceSynchronize());
  double gpuTime = 0;  // timer0.read();
#ifdef VERBOSE
  printf("FindExtrema time =            %.2f ms\n", gpuTime);
#endif
  return gpuTime;
}

void ClearPoints() {
  int totPts = 0;
  safeCall(hipMemcpyToSymbolAsync(HIP_SYMBOL(d_PointCounter), &totPts, sizeof(int)));
}

int GetPoints(std::vector<cv::KeyPoint> &h_pts, cv::KeyPoint *d_pts) {
  int numPts = 0;
  safeCall(hipMemcpyFromSymbolAsync(&numPts, HIP_SYMBOL(d_PointCounter), sizeof(int)));
  h_pts.resize(numPts);
  safeCall(hipMemcpy((float *)&h_pts[0], d_pts, sizeof(cv::KeyPoint) * numPts,
                      hipMemcpyDeviceToHost));
  return numPts;
}

__global__ void ExtractDescriptors(cv::KeyPoint *d_pts, CudaImage *d_imgs,
                                   float *_vals, int size2, int size3,
                                   int size4) {
  __shared__ float acc_vals[3 * 30 * EXTRACT_S];

  float *acc_vals_im = &acc_vals[0];
  float *acc_vals_dx = &acc_vals[30 * EXTRACT_S];
  float *acc_vals_dy = &acc_vals[2 * 30 * EXTRACT_S];

  int p = blockIdx.x;

  float *vals = &_vals[p * 3 * 29];

  float iratio = 1.0f / (1 << d_pts[p].octave);
  int scale = (int)(0.5f * d_pts[p].size * iratio + 0.5f);
  float xf = d_pts[p].pt.x * iratio;
  float yf = d_pts[p].pt.y * iratio;
  float ang = d_pts[p].angle;
  float co = cos(ang);
  float si = sin(ang);
  int tx = threadIdx.x;
  int lev = d_pts[p].class_id;
  float *imd = d_imgs[4 * lev + 0].d_data;
  float *dxd = d_imgs[4 * lev + 2].d_data;
  float *dyd = d_imgs[4 * lev + 3].d_data;
  int pitch = d_imgs[4 * lev + 0].pitch;
  int winsize = max(3 * size3, 4 * size4);

  for (int i = 0; i < 30; ++i) {
    acc_vals_im[i * EXTRACT_S + tx] = 0.f;
    acc_vals_dx[i * EXTRACT_S + tx] = 0.f;
    acc_vals_dy[i * EXTRACT_S + tx] = 0.f;
  }

  __syncthreads();

  for (int i = tx; i < winsize * winsize; i += EXTRACT_S) {
    int y = i / winsize;
    int x = i - winsize * y;
    int m = max(x, y);
    if (m >= winsize) continue;
    int l = x - size2;
    int k = y - size2;
    int xp = (int)(xf + scale * (k * co - l * si) + 0.5f);
    int yp = (int)(yf + scale * (k * si + l * co) + 0.5f);
    int pos = yp * pitch + xp;
    float im = imd[pos];
    float dx = dxd[pos];
    float dy = dyd[pos];
    float rx = -dx * si + dy * co;
    float ry = dx * co + dy * si;

    if (m < 2 * size2) {
      int x2 = (x < size2 ? 0 : 1);
      int y2 = (y < size2 ? 0 : 1);
      // atomicAdd(norm2, (x < size2 && y < size2 ? 1 : 0));
      // Add 2x2
      acc_vals[3 * (y2 * 2 + x2) + 3 * 30 * tx] += im;
      acc_vals[3 * (y2 * 2 + x2) + 3 * 30 * tx + 1] += rx;
      acc_vals[3 * (y2 * 2 + x2) + 3 * 30 * tx + 2] += ry;
    }
    if (m < 3 * size3) {
      int x3 = (x < size3 ? 0 : (x < 2 * size3 ? 1 : 2));
      int y3 = (y < size3 ? 0 : (y < 2 * size3 ? 1 : 2));
      // atomicAdd(norm3, (x < size3 && y < size3 ? 1 : 0));
      // Add 3x3
      acc_vals[3 * (4 + y3 * 3 + x3) + 3 * 30 * tx] += im;
      acc_vals[3 * (4 + y3 * 3 + x3) + 3 * 30 * tx + 1] += rx;
      acc_vals[3 * (4 + y3 * 3 + x3) + 3 * 30 * tx + 2] += ry;
    }
    if (m < 4 * size4) {
      int x4 = (x < 2 * size4 ? (x < size4 ? 0 : 1) : (x < 3 * size4 ? 2 : 3));
      int y4 = (y < 2 * size4 ? (y < size4 ? 0 : 1) : (y < 3 * size4 ? 2 : 3));
      // atomicAdd(norm4, (x < size4 && y < size4 ? 1 : 0));
      // Add 4x4
      acc_vals[3 * (4 + 9 + y4 * 4 + x4) + 3 * 30 * tx] += im;
      acc_vals[3 * (4 + 9 + y4 * 4 + x4) + 3 * 30 * tx + 1] += rx;
      acc_vals[3 * (4 + 9 + y4 * 4 + x4) + 3 * 30 * tx + 2] += ry;
    }
  }

  __syncthreads();

// Reduce stuff
#pragma unroll
  float acc_reg;
  for (int i = 0; i < 15; ++i) {
    // 0..31 takes care of even accs, 32..63 takes care of odd accs
    int offset = 2 * i + (tx < 32 ? 0 : 1);
    int tx_d = tx < 32 ? tx : tx - 32;
    for (int d = 0; d < 90; d += 30) {
      if (tx_d < 32) {
        acc_reg = acc_vals[3 * 30 * tx_d + offset + d] +
                  acc_vals[3 * 30 * (tx_d + 32) + offset + d];
        acc_reg += __shfl_down(acc_reg, 1);
        acc_reg += __shfl_down(acc_reg, 2);
        acc_reg += __shfl_down(acc_reg, 4);
        acc_reg += __shfl_down(acc_reg, 8);
        acc_reg += __shfl_down(acc_reg, 16);
      }
      if (tx_d == 0) {
        acc_vals[offset + d] = acc_reg;
      }
    }
  }

  __syncthreads();

  // Have 29*3 values to store
  // They are in acc_vals[0..28,64*30..64*30+28,64*60..64*60+28]
  if (tx < 29) {
    vals[tx] = acc_vals[tx];
    vals[29 + tx] = acc_vals[29 + tx];
    vals[2 * 29 + tx] = acc_vals[2 * 29 + tx];
  }
}

__global__ void ExtractDescriptors_serial(cv::KeyPoint *d_pts,
                                          CudaImage *d_imgs, float *_vals,
                                          int size2, int size3, int size4) {
  __shared__ float acc_vals[30 * EXTRACT_S];
  __shared__ float final_vals[3 * 30];

  int p = blockIdx.x;

  float *vals = &_vals[p * 3 * 29];

  float iratio = 1.0f / (1 << d_pts[p].octave);
  int scale = (int)(0.5f * d_pts[p].size * iratio + 0.5f);
  float xf = d_pts[p].pt.x * iratio;
  float yf = d_pts[p].pt.y * iratio;
  float ang = d_pts[p].angle;
  float co = cos(ang);
  float si = sin(ang);
  int tx = threadIdx.x;
  int lev = d_pts[p].class_id;
  float *imd = d_imgs[4 * lev + 0].d_data;
  float *dxd = d_imgs[4 * lev + 2].d_data;
  float *dyd = d_imgs[4 * lev + 3].d_data;
  int pitch = d_imgs[4 * lev + 0].pitch;
  int winsize = max(3 * size3, 4 * size4);

  // IM
  for (int i = 0; i < 30; ++i) {
    acc_vals[i * EXTRACT_S + tx] = 0.f;
  }

  __syncthreads();

  for (int i = tx; i < winsize * winsize; i += EXTRACT_S) {
    int y = i / winsize;
    int x = i - winsize * y;
    int m = max(x, y);
    if (m >= winsize) continue;
    int l = x - size2;
    int k = y - size2;
    int xp = (int)(xf + scale * (k * co - l * si) + 0.5f);
    int yp = (int)(yf + scale * (k * si + l * co) + 0.5f);
    int pos = yp * pitch + xp;
    float im = imd[pos];
    if (m < 2 * size2) {
      int x2 = (x < size2 ? 0 : 1);
      int y2 = (y < size2 ? 0 : 1);
      // atomicAdd(norm2, (x < size2 && y < size2 ? 1 : 0));
      // Add 2x2
      acc_vals[(y2 * 2 + x2) + 30 * tx] += im;
    }
    if (m < 3 * size3) {
      int x3 = (x < size3 ? 0 : (x < 2 * size3 ? 1 : 2));
      int y3 = (y < size3 ? 0 : (y < 2 * size3 ? 1 : 2));
      // atomicAdd(norm3, (x < size3 && y < size3 ? 1 : 0));
      // Add 3x3
      acc_vals[(4 + y3 * 3 + x3) + 30 * tx] += im;
    }
    if (m < 4 * size4) {
      int x4 = (x < 2 * size4 ? (x < size4 ? 0 : 1) : (x < 3 * size4 ? 2 : 3));
      int y4 = (y < 2 * size4 ? (y < size4 ? 0 : 1) : (y < 3 * size4 ? 2 : 3));
      // atomicAdd(norm4, (x < size4 && y < size4 ? 1 : 0));
      // Add 4x4
      acc_vals[(4 + 9 + y4 * 4 + x4) + 30 * tx] += im;
    }
  }

  __syncthreads();

// Reduce stuff
#pragma unroll
  for (int i = 0; i < 15; ++i) {
    // 0..31 takes care of even accs, 32..63 takes care of odd accs
    int offset = 2 * i + (tx < 32 ? 0 : 1);
    int tx_d = tx < 32 ? tx : tx - 32;
    int acc_idx = 30 * tx_d + offset;
    if (tx_d < 32) {
      acc_vals[acc_idx] += acc_vals[acc_idx + 30 * 32];
    }
    if (tx_d < 16) {
      acc_vals[acc_idx] += acc_vals[acc_idx + 30 * 16];
    }
    if (tx_d < 8) {
      acc_vals[acc_idx] += acc_vals[acc_idx + 30 * 8];
    }
    if (tx_d < 4) {
      acc_vals[acc_idx] += acc_vals[acc_idx + 30 * 4];
    }
    if (tx_d < 2) {
      acc_vals[acc_idx] += acc_vals[acc_idx + 30 * 2];
    }
    if (tx_d < 1) {
      final_vals[3 * offset] = acc_vals[acc_idx] + acc_vals[offset + 30];
    }
  }

  // DX
  for (int i = 0; i < 30; ++i) {
    acc_vals[i * EXTRACT_S + tx] = 0.f;
  }

  __syncthreads();

  for (int i = tx; i < winsize * winsize; i += EXTRACT_S) {
    int y = i / winsize;
    int x = i - winsize * y;
    int m = max(x, y);
    if (m >= winsize) continue;
    int l = x - size2;
    int k = y - size2;
    int xp = (int)(xf + scale * (k * co - l * si) + 0.5f);
    int yp = (int)(yf + scale * (k * si + l * co) + 0.5f);
    int pos = yp * pitch + xp;
    float dx = dxd[pos];
    float dy = dyd[pos];
    float rx = -dx * si + dy * co;
    if (m < 2 * size2) {
      int x2 = (x < size2 ? 0 : 1);
      int y2 = (y < size2 ? 0 : 1);
      // atomicAdd(norm2, (x < size2 && y < size2 ? 1 : 0));
      // Add 2x2
      acc_vals[(y2 * 2 + x2) + 30 * tx] += rx;
    }
    if (m < 3 * size3) {
      int x3 = (x < size3 ? 0 : (x < 2 * size3 ? 1 : 2));
      int y3 = (y < size3 ? 0 : (y < 2 * size3 ? 1 : 2));
      // atomicAdd(norm3, (x < size3 && y < size3 ? 1 : 0));
      // Add 3x3
      acc_vals[(4 + y3 * 3 + x3) + 30 * tx] += rx;
    }
    if (m < 4 * size4) {
      int x4 = (x < 2 * size4 ? (x < size4 ? 0 : 1) : (x < 3 * size4 ? 2 : 3));
      int y4 = (y < 2 * size4 ? (y < size4 ? 0 : 1) : (y < 3 * size4 ? 2 : 3));
      // atomicAdd(norm4, (x < size4 && y < size4 ? 1 : 0));
      // Add 4x4
      acc_vals[(4 + 9 + y4 * 4 + x4) + 30 * tx] += rx;
    }
  }

  __syncthreads();

// Reduce stuff
#pragma unroll
  for (int i = 0; i < 15; ++i) {
    // 0..31 takes care of even accs, 32..63 takes care of odd accs
    int offset = 2 * i + (tx < 32 ? 0 : 1);
    int tx_d = tx < 32 ? tx : tx - 32;
    int acc_idx = 30 * tx_d + offset;
    if (tx_d < 32) {
      acc_vals[acc_idx] += acc_vals[acc_idx + 30 * 32];
    }
    if (tx_d < 16) {
      acc_vals[acc_idx] += acc_vals[acc_idx + 30 * 16];
    }
    if (tx_d < 8) {
      acc_vals[acc_idx] += acc_vals[acc_idx + 30 * 8];
    }
    if (tx_d < 4) {
      acc_vals[acc_idx] += acc_vals[acc_idx + 30 * 4];
    }
    if (tx_d < 2) {
      acc_vals[acc_idx] += acc_vals[acc_idx + 30 * 2];
    }
    if (tx_d < 1) {
      final_vals[3 * offset] = acc_vals[acc_idx] + acc_vals[offset + 30];
    }
  }

  // DY
  for (int i = 0; i < 30; ++i) {
    acc_vals[i * EXTRACT_S + tx] = 0.f;
  }

  __syncthreads();

  for (int i = tx; i < winsize * winsize; i += EXTRACT_S) {
    int y = i / winsize;
    int x = i - winsize * y;
    int m = max(x, y);
    if (m >= winsize) continue;
    int l = x - size2;
    int k = y - size2;
    int xp = (int)(xf + scale * (k * co - l * si) + 0.5f);
    int yp = (int)(yf + scale * (k * si + l * co) + 0.5f);
    int pos = yp * pitch + xp;
    float dx = dxd[pos];
    float dy = dyd[pos];
    float ry = dx * co + dy * si;
    if (m < 2 * size2) {
      int x2 = (x < size2 ? 0 : 1);
      int y2 = (y < size2 ? 0 : 1);
      // atomicAdd(norm2, (x < size2 && y < size2 ? 1 : 0));
      // Add 2x2
      acc_vals[(y2 * 2 + x2) + 30 * tx] += ry;
    }
    if (m < 3 * size3) {
      int x3 = (x < size3 ? 0 : (x < 2 * size3 ? 1 : 2));
      int y3 = (y < size3 ? 0 : (y < 2 * size3 ? 1 : 2));
      // atomicAdd(norm3, (x < size3 && y < size3 ? 1 : 0));
      // Add 3x3
      acc_vals[(4 + y3 * 3 + x3) + 30 * tx] += ry;
    }
    if (m < 4 * size4) {
      int x4 = (x < 2 * size4 ? (x < size4 ? 0 : 1) : (x < 3 * size4 ? 2 : 3));
      int y4 = (y < 2 * size4 ? (y < size4 ? 0 : 1) : (y < 3 * size4 ? 2 : 3));
      // atomicAdd(norm4, (x < size4 && y < size4 ? 1 : 0));
      // Add 4x4
      acc_vals[(4 + 9 + y4 * 4 + x4) + 30 * tx] += ry;
    }
  }

  __syncthreads();

// Reduce stuff
#pragma unroll
  for (int i = 0; i < 15; ++i) {
    // 0..31 takes care of even accs, 32..63 takes care of odd accs
    int offset = 2 * i + (tx < 32 ? 0 : 1);
    int tx_d = tx < 32 ? tx : tx - 32;
    int acc_idx = 30 * tx_d + offset;
    if (tx_d < 32) {
      acc_vals[acc_idx] += acc_vals[acc_idx + 30 * 32];
    }
    if (tx_d < 16) {
      acc_vals[acc_idx] += acc_vals[acc_idx + 30 * 16];
    }
    if (tx_d < 8) {
      acc_vals[acc_idx] += acc_vals[acc_idx + 30 * 8];
    }
    if (tx_d < 4) {
      acc_vals[acc_idx] += acc_vals[acc_idx + 30 * 4];
    }
    if (tx_d < 2) {
      acc_vals[acc_idx] += acc_vals[acc_idx + 30 * 2];
    }
    if (tx_d < 1) {
      final_vals[3 * offset] = acc_vals[acc_idx] + acc_vals[offset + 30];
    }
  }

  __syncthreads();

  // Have 29*3 values to store
  // They are in acc_vals[0..28,64*30..64*30+28,64*60..64*60+28]
  if (tx < 29) {
    vals[tx] = final_vals[tx];
    vals[29 + tx] = final_vals[29 + tx];
    vals[2 * 29 + tx] = final_vals[2 * 29 + tx];
  }
}

__global__ void ExtractDescriptors_atomic_shared(cv::KeyPoint *d_pts,
                                                 CudaImage *d_imgs,
                                                 float *_vals, int size2,
                                                 int size3, int size4) {
  __shared__ float acc_vals[3 * 30 * EXTRACT_S / 2];

  float *acc_vals_im = &acc_vals[0];
  float *acc_vals_dx = &acc_vals[30 * EXTRACT_S / 2];
  float *acc_vals_dy = &acc_vals[2 * 30 * EXTRACT_S / 2];

  int p = blockIdx.x;

  float *vals = &_vals[p * 3 * 29];

  float iratio = 1.0f / (1 << d_pts[p].octave);
  int scale = (int)(0.5f * d_pts[p].size * iratio + 0.5f);
  float xf = d_pts[p].pt.x * iratio;
  float yf = d_pts[p].pt.y * iratio;
  float ang = d_pts[p].angle;
  float co = cos(ang);
  float si = sin(ang);
  int tx = threadIdx.x;
  int lev = d_pts[p].class_id;
  float *imd = d_imgs[4 * lev + 0].d_data;
  float *dxd = d_imgs[4 * lev + 2].d_data;
  float *dyd = d_imgs[4 * lev + 3].d_data;
  int pitch = d_imgs[4 * lev + 0].pitch;
  int winsize = max(3 * size3, 4 * size4);

  for (int i = 0; i < 15; ++i) {
    acc_vals_im[i * EXTRACT_S + tx] = 0.f;
    acc_vals_dx[i * EXTRACT_S + tx] = 0.f;
    acc_vals_dy[i * EXTRACT_S + tx] = 0.f;
  }

  __syncthreads();

  for (int i = tx; i < winsize * winsize; i += EXTRACT_S) {
    int y = i / winsize;
    int x = i - winsize * y;
    int m = max(x, y);
    if (m >= winsize) continue;
    int l = x - size2;
    int k = y - size2;
    int xp = (int)(xf + scale * (k * co - l * si) + 0.5f);
    int yp = (int)(yf + scale * (k * si + l * co) + 0.5f);
    int pos = yp * pitch + xp;
    float im = imd[pos];
    float dx = dxd[pos];
    float dy = dyd[pos];
    float rx = -dx * si + dy * co;
    float ry = dx * co + dy * si;

    int tx_d = tx & (EXTRACT_S >> 1 - 1);
    if (m < 2 * size2) {
      int x2 = (x < size2 ? 0 : 1);
      int y2 = (y < size2 ? 0 : 1);
      // atomicAdd(norm2, (x < size2 && y < size2 ? 1 : 0));
      // Add 2x2
      atomicAdd(&acc_vals[3 * (y2 * 2 + x2) + 3 * 30 * tx_d], im);
      atomicAdd(&acc_vals[3 * (y2 * 2 + x2) + 3 * 30 * tx_d + 1], rx);
      atomicAdd(&acc_vals[3 * (y2 * 2 + x2) + 3 * 30 * tx_d + 2], ry);
    }
    if (m < 3 * size3) {
      int x3 = (x < size3 ? 0 : (x < 2 * size3 ? 1 : 2));
      int y3 = (y < size3 ? 0 : (y < 2 * size3 ? 1 : 2));
      // atomicAdd(norm3, (x < size3 && y < size3 ? 1 : 0));
      // Add 3x3
      atomicAdd(&acc_vals[3 * (4 + y3 * 3 + x3) + 3 * 30 * tx_d], im);
      atomicAdd(&acc_vals[3 * (4 + y3 * 3 + x3) + 3 * 30 * tx_d + 1], rx);
      atomicAdd(&acc_vals[3 * (4 + y3 * 3 + x3) + 3 * 30 * tx_d + 2], ry);
    }
    if (m < 4 * size4) {
      int x4 = (x < 2 * size4 ? (x < size4 ? 0 : 1) : (x < 3 * size4 ? 2 : 3));
      int y4 = (y < 2 * size4 ? (y < size4 ? 0 : 1) : (y < 3 * size4 ? 2 : 3));
      // atomicAdd(norm4, (x < size4 && y < size4 ? 1 : 0));
      // Add 4x4
      atomicAdd(&acc_vals[3 * (4 + 9 + y4 * 4 + x4) + 3 * 30 * tx_d], im);
      atomicAdd(&acc_vals[3 * (4 + 9 + y4 * 4 + x4) + 3 * 30 * tx_d + 1], rx);
      atomicAdd(&acc_vals[3 * (4 + 9 + y4 * 4 + x4) + 3 * 30 * tx_d + 2], ry);
    }
  }

  __syncthreads();

// Reduce stuff
#pragma unroll
  for (int i = 0; i < 15; ++i) {
    // 0..31 takes care of even accs, 32..63 takes care of odd accs
    int offset = 2 * i + (tx < 32 ? 0 : 1);
    int tx_d = tx < 32 ? tx : tx - 32;
    /*if (tx_d < 32) {
      acc_vals[3*30 * tx_d/2 + offset] += acc_vals[3*30/2 * (tx_d + 32) +
    offset];
      acc_vals[3*30 * tx_d/2 + offset+30] += acc_vals[3*30/2 * (tx_d + 32) +
    offset+30];
      acc_vals[3*30 * tx_d/2 + offset+60] += acc_vals[3*30/2 * (tx_d + 32) +
    offset+60];
    }*/
    if (tx_d < 16) {
      acc_vals[3 * 30 * tx_d + offset] +=
          acc_vals[3 * 30 * (tx_d + 16) + offset];
      acc_vals[3 * 30 * tx_d + offset + 30] +=
          acc_vals[3 * 30 * (tx_d + 16) + offset + 30];
      acc_vals[3 * 30 * tx_d + offset + 60] +=
          acc_vals[3 * 30 * (tx_d + 16) + offset + 60];
    }
    if (tx_d < 8) {
      acc_vals[3 * 30 * tx_d + offset] +=
          acc_vals[3 * 30 * (tx_d + 8) + offset];
      acc_vals[3 * 30 * tx_d + offset + 30] +=
          acc_vals[3 * 30 * (tx_d + 8) + offset + 30];
      acc_vals[3 * 30 * tx_d + offset + 60] +=
          acc_vals[3 * 30 * (tx_d + 8) + offset + 60];
    }
    if (tx_d < 4) {
      acc_vals[3 * 30 * tx_d + offset] +=
          acc_vals[3 * 30 * (tx_d + 4) + offset];
      acc_vals[3 * 30 * tx_d + offset + 30] +=
          acc_vals[3 * 30 * (tx_d + 4) + offset + 30];
      acc_vals[3 * 30 * tx_d + offset + 60] +=
          acc_vals[3 * 30 * (tx_d + 4) + offset + 60];
    }
    if (tx_d < 2) {
      acc_vals[3 * 30 * tx_d + offset] +=
          acc_vals[3 * 30 * (tx_d + 2) + offset];
      acc_vals[3 * 30 * tx_d + offset + 30] +=
          acc_vals[3 * 30 * (tx_d + 2) + offset + 30];
      acc_vals[3 * 30 * tx_d + offset + 60] +=
          acc_vals[3 * 30 * (tx_d + 2) + offset + 60];
    }
    if (tx_d < 1) {
      acc_vals[offset] += acc_vals[3 * 30 + offset];
      acc_vals[offset + 30] += acc_vals[3 * 30 + offset + 30];
      acc_vals[offset + 60] += acc_vals[3 * 30 + offset + 60];
    }
  }

  __syncthreads();

  // Have 29*3 values to store
  // They are in acc_vals[0..28,64*30..64*30+28,64*60..64*60+28]
  if (tx < 29) {
    vals[tx] = acc_vals[tx];
    vals[29 + tx] = acc_vals[29 + tx];
    vals[2 * 29 + tx] = acc_vals[2 * 29 + tx];
  }
}

__global__ void ExtractDescriptors_atomic(cv::KeyPoint *d_pts,
                                          CudaImage *d_imgs, float *_vals,
                                          int size2, int size3, int size4) {
  __shared__ float acc_vals[3 * 29];

  float *acc_vals_im = &acc_vals[0];
  float *acc_vals_dx = &acc_vals[29];
  float *acc_vals_dy = &acc_vals[2 * 29];

  int p = blockIdx.x;

  float *vals = &_vals[p * 3 * 29];

  float iratio = 1.0f / (1 << d_pts[p].octave);
  int scale = (int)(0.5f * d_pts[p].size * iratio + 0.5f);
  float xf = d_pts[p].pt.x * iratio;
  float yf = d_pts[p].pt.y * iratio;
  float ang = d_pts[p].angle;
  float co = cos(ang);
  float si = sin(ang);
  int tx = threadIdx.x;
  int lev = d_pts[p].class_id;
  float *imd = d_imgs[4 * lev + 0].d_data;
  float *dxd = d_imgs[4 * lev + 2].d_data;
  float *dyd = d_imgs[4 * lev + 3].d_data;
  int pitch = d_imgs[4 * lev + 0].pitch;
  int winsize = max(3 * size3, 4 * size4);

  if (tx < 29) {
    acc_vals_im[tx] = 0.f;
    acc_vals_dx[tx] = 0.f;
    acc_vals_dy[tx] = 0.f;
  }

  __syncthreads();

  for (int i = tx; i < winsize * winsize; i += EXTRACT_S) {
    int y = i / winsize;
    int x = i - winsize * y;
    int m = max(x, y);
    if (m >= winsize) continue;
    int l = x - size2;
    int k = y - size2;
    int xp = (int)(xf + scale * (k * co - l * si) + 0.5f);
    int yp = (int)(yf + scale * (k * si + l * co) + 0.5f);
    int pos = yp * pitch + xp;
    float im = imd[pos];
    float dx = dxd[pos];
    float dy = dyd[pos];
    float rx = -dx * si + dy * co;
    float ry = dx * co + dy * si;

    if (m < 2 * size2) {
      int x2 = (x < size2 ? 0 : 1);
      int y2 = (y < size2 ? 0 : 1);
      // atomicAdd(norm2, (x < size2 && y < size2 ? 1 : 0));
      // Add 2x2
      atomicAdd(&acc_vals[3 * (y2 * 2 + x2)], im);
      atomicAdd(&acc_vals[3 * (y2 * 2 + x2) + 1], rx);
      atomicAdd(&acc_vals[3 * (y2 * 2 + x2) + 2], ry);
    }
    if (m < 3 * size3) {
      int x3 = (x < size3 ? 0 : (x < 2 * size3 ? 1 : 2));
      int y3 = (y < size3 ? 0 : (y < 2 * size3 ? 1 : 2));
      // atomicAdd(norm3, (x < size3 && y < size3 ? 1 : 0));
      // Add 3x3
      atomicAdd(&acc_vals[3 * (4 + y3 * 3 + x3)], im);
      atomicAdd(&acc_vals[3 * (4 + y3 * 3 + x3) + 1], rx);
      atomicAdd(&acc_vals[3 * (4 + y3 * 3 + x3) + 2], ry);
    }
    if (m < 4 * size4) {
      int x4 = (x < 2 * size4 ? (x < size4 ? 0 : 1) : (x < 3 * size4 ? 2 : 3));
      int y4 = (y < 2 * size4 ? (y < size4 ? 0 : 1) : (y < 3 * size4 ? 2 : 3));
      // atomicAdd(norm4, (x < size4 && y < size4 ? 1 : 0));
      // Add 4x4
      atomicAdd(&acc_vals[3 * (4 + 9 + y4 * 4 + x4)], im);
      atomicAdd(&acc_vals[3 * (4 + 9 + y4 * 4 + x4) + 1], rx);
      atomicAdd(&acc_vals[3 * (4 + 9 + y4 * 4 + x4) + 2], ry);
    }
  }

  __syncthreads();

  // Have 29*3 values to store
  // They are in acc_vals[0..28,64*30..64*30+28,64*60..64*60+28]
  if (tx < 29) {
    vals[tx] = acc_vals[tx];
    vals[29 + tx] = acc_vals[29 + tx];
    vals[2 * 29 + tx] = acc_vals[2 * 29 + tx];
  }
}

__global__ void BuildDescriptor(float *_valsim, unsigned char *_desc) {
  int p = blockIdx.x;

  size_t idx = threadIdx.x;

  if (idx < 61) {
    float *valsim = &_valsim[3 * 29 * p];

    unsigned char *desc = &_desc[61 * p];

    unsigned char desc_r = 0;

#pragma unroll
    for (int i = 0; i < (idx == 60 ? 6 : 8); ++i) {
      int idx1 = comp_idx_1[idx * 8 + i];
      int idx2 = comp_idx_2[idx * 8 + i];
      desc_r |= (valsim[idx1] > valsim[idx2] ? 1 : 0) << i;
    }

    desc[idx] = desc_r;
  }
}

#define NTHREADS_MATCH 32
__global__ void MatchDescriptors(unsigned char *d1, unsigned char *d2,
                                 int pitch, int nkpts_2, cv::DMatch *matches) {
  int p = blockIdx.x;

  int x = threadIdx.x;

  __shared__ int idxBest[NTHREADS_MATCH];
  __shared__ int idxSecondBest[NTHREADS_MATCH];
  __shared__ int scoreBest[NTHREADS_MATCH];
  __shared__ int scoreSecondBest[NTHREADS_MATCH];

  idxBest[x] = 0;
  idxSecondBest[x] = 0;
  scoreBest[x] = 512;
  scoreSecondBest[x] = 512;

  __syncthreads();

  // working single threaded version, the problem was popcll
  //  int idxBest = 1;
  //  int idxSecondBest = 0;
  //  int scoreBest = 512;
  //  int scoreSecondBest = 512;

  //  for (int i = 0; i < nkpts_2; ++i) {
  //    int score = 0;
  //    for (int j = 0; j < 64; ++j) {
  //      score += __popc(d1[pitch * p + j] ^ d2[pitch * i + j]);
  //    }

  //    if (score < scoreBest) {
  //      scoreSecondBest = scoreBest;
  //      scoreBest = score;
  //      idxSecondBest = idxBest;
  //      idxBest = i;

  //    } else if (score < scoreSecondBest) {
  //      scoreSecondBest = score;
  //      idxSecondBest = i;
  //    }
  //  }

  //    int idxBest = 1;
  //    int idxSecondBest = 0;
  //    int scoreBest = 512;
  //    int scoreSecondBest = 512;

  //    for (int i=0; i<nkpts_2; i+=32) {
  //        if( i+x < nkpts_2) {
  //            // Check d1[p] with d2[i]
  //            int score = 0;
  //            for(int j=0; j<16; ++j) {
  //                score += __popcll(d1[pitch*p+4*j] ^ d2[pitch*(i+x)+4*j]);
  //            }
  //            if( score < scoreBest ) {
  //                scoreSecondBest = scoreBest;
  //                scoreBest = score;
  //                idxSecondBest = idxBest;
  //                idxBest = i+x;
  //            } else if( score < scoreSecondBest ) {
  //                scoreSecondBest = score;
  //                idxSecondBest = i+x;
  //            }
  //        }
  //    }

  // curent version fixed with popc, still not convinced
  unsigned long long *d1i = (unsigned long long*)(d1 + pitch*p);

  for (int i = 0; i < nkpts_2; i += NTHREADS_MATCH) {
    unsigned long long *d2i = (unsigned long long*)(d2 + pitch*(x+i));
    if (i + x < nkpts_2) {
      // Check d1[p] with d2[i]
      int score = 0;
      for (int j = 0; j < 8; ++j) {
        score += __popcll(d1i[j] ^ d2i[j]);
      }
      if (score < scoreBest[x]) {
        scoreSecondBest[x] = scoreBest[x];
        scoreBest[x] = score;
        idxSecondBest[x] = idxBest[x];
        idxBest[x] = i + x;
      } else if (score < scoreSecondBest[x]) {
        scoreSecondBest[x] = score;
        idxSecondBest[x] = i + x;
      }
    }
  }

  //    for( int i=16; i>=1; i/=2) {
  //        int tBest = __shfl_down(scoreBest,i);
  //        int tIdx = __shfl_down(idxBest,i);
  //        if(tBest < scoreBest) {
  //            scoreSecondBest = scoreBest;
  //            idxSecondBest = idxBest;
  //            scoreBest = tBest;
  //            idxBest = tIdx;
  //        }
  //        tBest = __shfl_down(scoreSecondBest,i);
  //        tIdx = __shfl_down(idxSecondBest,i);
  //        if(tBest < scoreSecondBest) {
  //            scoreSecondBest = tBest;
  //            idxSecondBest = tIdx;
  //        }
  //    }

  __syncthreads();

  for (int i = NTHREADS_MATCH / 2; i >= 1; i /= 2) {
    if (x < i) {
      if (scoreBest[x + i] < scoreBest[x]) {
        scoreSecondBest[x] = scoreBest[x];
        scoreBest[x] = scoreBest[x + i];
        idxSecondBest[x] = idxBest[x];
        idxBest[x] = idxBest[x + i];
      } else if (scoreBest[x + i] < scoreSecondBest[x]) {
        scoreSecondBest[x] = scoreBest[x + i];
        idxSecondBest[x] = idxBest[x + i];
      }
      if (scoreSecondBest[x + i] < scoreSecondBest[x]) {
        scoreSecondBest[x] = scoreSecondBest[x + i];
        idxSecondBest[x] = idxSecondBest[x + i];
      }
    }
    }
    // if(i>16) __syncthreads();
    //        if(x<i) {
    //            if( scoreBest[x+i] < scoreSecondBest[x] ) {
    //                scoreSecondBest[x] = scoreBest[x+i];
    //                idxSecondBest[x] = idxBest[x+i];
    //            } else if (scoreSecondBest[x+i] < scoreSecondBest[x] ) {
    //                scoreSecondBest[x] = scoreSecondBest[x+i];
    //                idxSecondBest[x] = idxSecondBest[x+i];
    //            }
    //        }
    //        if(i>16) __syncthreads();
    //}

    /*for (int i = 1; i <= NTHREADS_MATCH; ++i) {
      if (scoreBest[i] < scoreBest[0]) {
        scoreSecondBest[0] = scoreBest[0];
        scoreBest[0] = scoreBest[i];
        idxSecondBest[0] = idxBest[0];
        idxBest[0] = idxBest[i];
      }  else if( scoreBest[i] < scoreSecondBest[0] ) {
           scoreSecondBest[0] = scoreBest[i];
           idxSecondBest[0] = idxBest[i];
       }
       if(scoreSecondBest[i] < scoreSecondBest[0]) {
           scoreSecondBest[0] = scoreSecondBest[i];
           idxSecondBest[0] = idxSecondBest[i];
       }
    }*/

    //    if(x==0) {
    //        matches[2*p].queryIdx = p;
    //        matches[2*p].trainIdx = idxBest;
    //        matches[2*p].distance = scoreBest;
    //        matches[2*p+1].queryIdx = p;
    //        matches[2*p+1].trainIdx = idxSecondBest;
    //        matches[2*p+1].distance = scoreSecondBest;
    //    }

    if (x == 0) {
      matches[2 * p].queryIdx = p;
      matches[2 * p].trainIdx = idxBest[x];
      matches[2 * p].distance = scoreBest[x];
      matches[2 * p + 1].queryIdx = p;
      matches[2 * p + 1].trainIdx = idxSecondBest[x];
      matches[2 * p + 1].distance = scoreSecondBest[x];
    }
  }

  void MatchDescriptors(cv::Mat & desc_query, cv::Mat & desc_train,
                        std::vector<std::vector<cv::DMatch> > & dmatches) {
    size_t pitch1, pitch2;
    unsigned char *descq_d;
    hipMallocPitch(&descq_d, &pitch1, 64, desc_query.rows);
    hipMemset2D(descq_d, pitch1, 0, 64, desc_query.rows);
    hipMemcpy2D(descq_d, pitch1, desc_query.data, desc_query.cols,
                 desc_query.cols, desc_query.rows, hipMemcpyHostToDevice);
    unsigned char *desct_d;
    hipMallocPitch(&desct_d, &pitch2, 64, desc_train.rows);
    hipMemset2D(desct_d, pitch2, 0, 64, desc_train.rows);
    hipMemcpy2D(desct_d, pitch2, desc_train.data, desc_train.cols,
                 desc_train.cols, desc_train.rows, hipMemcpyHostToDevice);

    dim3 block(desc_query.rows);

    cv::DMatch *dmatches_d;
    hipMalloc(&dmatches_d, desc_query.rows * 2 * sizeof(cv::DMatch));

    MatchDescriptors << <block, NTHREADS_MATCH>>>
        (descq_d, desct_d, pitch1, desc_train.rows, dmatches_d);

    cv::DMatch *dmatches_h = new cv::DMatch[2 * desc_query.rows];
    hipMemcpy(dmatches_h, dmatches_d, desc_query.rows * 2 * sizeof(cv::DMatch),
               hipMemcpyDeviceToHost);

    for (int i = 0; i < desc_query.rows; ++i) {
      std::vector<cv::DMatch> tdmatch;
      tdmatch.push_back(dmatches_h[2 * i]);
      tdmatch.push_back(dmatches_h[2 * i + 1]);
      dmatches.push_back(tdmatch);
    }

    hipFree(descq_d);
    hipFree(desct_d);
    hipFree(dmatches_d);

    delete[] dmatches_h;
  }

  double ExtractDescriptors(std::vector<cv::KeyPoint> & h_pts,
                            cv::KeyPoint * d_pts,
                            std::vector<CudaImage> & h_imgs, CudaImage * d_imgs,
                            unsigned char *desc_h, int patsize) {
    int size2 = patsize;
    int size3 = ceil(2.0f * patsize / 3.0f);
    int size4 = ceil(0.5f * patsize);
    int numPts = h_pts.size();

    // TimerGPU timer0(0);
    dim3 blocks(numPts);
    dim3 threads(EXTRACT_S);

    float *vals_h = new float[3 * 29 * numPts];
    float *vals_d;
    hipMalloc(&vals_d, 3 * 29 * numPts * sizeof(float));

    ExtractDescriptors << <blocks, threads>>>
        (d_pts, d_imgs, vals_d, size2, size3, size4);

//#define PRINT_KPT

#ifdef PRINT_KPT
    hipMemcpy(vals_h, vals_d, 3 * 29 * numPts * sizeof(float),
               hipMemcpyDeviceToHost);

    //  int xkpt = 865, ykpt = 30;
    int xkpt = 840, ykpt = 45;

    int idx = -1;
    for (int i = 0; i < h_pts.size(); ++i) {
      if ((int)h_pts[i].pt.x == xkpt && (int)h_pts[i].pt.y == ykpt) idx = i;
    }
#endif

    static int comp_idx_1_h[61 * 8];
    static int comp_idx_2_h[61 * 8];

    int cntr = 0;
    for (int j = 0; j < 4; ++j) {
      for (int i = j + 1; i < 4; ++i) {
        comp_idx_1_h[cntr] = 3 * j;
        comp_idx_2_h[cntr] = 3 * i;
        cntr++;
      }
    }
    for (int j = 0; j < 3; ++j) {
      for (int i = j + 1; i < 4; ++i) {
        comp_idx_1_h[cntr] = 3 * j + 1;
        comp_idx_2_h[cntr] = 3 * i + 1;
        cntr++;
      }
    }
    for (int j = 0; j < 3; ++j) {
      for (int i = j + 1; i < 4; ++i) {
        comp_idx_1_h[cntr] = 3 * j + 2;
        comp_idx_2_h[cntr] = 3 * i + 2;
        cntr++;
      }
    }

    // 3x3
    for (int j = 4; j < 12; ++j) {
      for (int i = j + 1; i < 13; ++i) {
        comp_idx_1_h[cntr] = 3 * j;
        comp_idx_2_h[cntr] = 3 * i;
        cntr++;
      }
    }
    for (int j = 4; j < 12; ++j) {
      for (int i = j + 1; i < 13; ++i) {
        comp_idx_1_h[cntr] = 3 * j + 1;
        comp_idx_2_h[cntr] = 3 * i + 1;
        cntr++;
      }
    }
    for (int j = 4; j < 12; ++j) {
      for (int i = j + 1; i < 13; ++i) {
        comp_idx_1_h[cntr] = 3 * j + 2;
        comp_idx_2_h[cntr] = 3 * i + 2;
        cntr++;
      }
    }

    // 4x4
    for (int j = 13; j < 28; ++j) {
      for (int i = j + 1; i < 29; ++i) {
        comp_idx_1_h[cntr] = 3 * j;
        comp_idx_2_h[cntr] = 3 * i;
        cntr++;
      }
    }
    for (int j = 13; j < 28; ++j) {
      for (int i = j + 1; i < 29; ++i) {
        comp_idx_1_h[cntr] = 3 * j + 1;
        comp_idx_2_h[cntr] = 3 * i + 1;
        cntr++;
      }
    }
    for (int j = 13; j < 28; ++j) {
      for (int i = j + 1; i < 29; ++i) {
        comp_idx_1_h[cntr] = 3 * j + 2;
        comp_idx_2_h[cntr] = 3 * i + 2;
        cntr++;
      }
    }

    hipMemcpyToSymbolAsync(HIP_SYMBOL(comp_idx_1), comp_idx_1_h, 8 * 61 * sizeof(int));
    hipMemcpyToSymbolAsync(HIP_SYMBOL(comp_idx_2), comp_idx_2_h, 8 * 61 * sizeof(int));

    unsigned char *desc_d;
    hipMalloc(&desc_d, numPts * 61);
    hipMemsetAsync(desc_d, 0, numPts * 61);
    BuildDescriptor << <blocks, 64>>> (vals_d, desc_d);

    hipMemcpy(desc_h, desc_d, 61 * numPts, hipMemcpyDeviceToHost);
#ifdef PRINT_KPT
    if (idx == -1)
      std::cout << "GPU error:: keypoint not found! " << std::endl;
    else {
      float sum2 = 0, sum3 = 0, sum4 = 0;
      for (int i = 0; i < 4; ++i) {
        sum2 += vals_h[3 * 29 * idx + 3 * i];
      }
      for (int i = 0; i < 9; ++i) {
        sum3 += vals_h[3 * 29 * idx + 12 + 3 * i];
      }
      for (int i = 0; i < 16; ++i) {
        sum4 += vals_h[3 * 29 * idx + 39 + 3 * i];
      }

      std::cout << "sums: " << sum2 << " " << sum3 << " " << sum4 << std::endl;

      std::cout << "Keypoint idx: " << idx << std::endl;

      std::cout << "GPU output:\n";
      std::cout << "2x2\n";
      for (int i = 0; i < 12; ++i) {
        std::cout << vals_h[3 * 29 * idx + i] << " ";
      }
      std::cout << std::endl;
      std::cout << "3x3";
      std::cout << std::endl;
      for (int i = 12; i < 39; ++i) {
        std::cout << vals_h[3 * 29 * idx + i] << " ";
      }
      std::cout << std::endl;
      std::cout << "4x4";
      std::cout << std::endl;
      for (int i = 39; i < 3 * 29; ++i) {
        std::cout << vals_h[3 * 29 * idx + i] << " ";
      }
      std::cout << std::endl;
      std::cout << "descriptor\n";
      std::cout << std::endl;
      for (int i = 0; i < 61; ++i) {
        std::cout << (unsigned int)desc_h[idx * 61 + i] << " ";
      }
      std::cout << "\n";
    }
#endif

    ////checkMsg("ExtractDescriptors() execution failed\n");
    // safeCall(hipDeviceSynchronize());
    double gpuTime = 0;  // timer0.read();
#ifdef VERBOSE
    printf("ExtractDescriptors time =     %.2f ms\n", gpuTime);
#endif

    hipFree(vals_d);
    hipFree(desc_d);

    delete[] vals_h;

    return gpuTime;
  }

  __global__ void FindOrientation(cv::KeyPoint * d_pts, CudaImage * d_imgs) {
    __shared__ float resx[42], resy[42];
    __shared__ float re8x[42], re8y[42];
    int p = blockIdx.x;
    int tx = threadIdx.x;
    if (tx < 48) resx[tx] = resy[tx] = 0.0f;
    __syncthreads();
    int lev = d_pts[p].class_id;
    float *dxd = d_imgs[4 * lev + 2].d_data;
    float *dyd = d_imgs[4 * lev + 3].d_data;
    int pitch = d_imgs[4 * lev + 0].pitch;
    int octave = d_pts[p].octave;
    int step = (int)(0.5f * d_pts[p].size + 0.5f) >> octave;
    int x = (int)(d_pts[p].pt.x + 0.5f) >> octave;
    int y = (int)(d_pts[p].pt.y + 0.5f) >> octave;
    int i = (tx & 15) - 6;
    int j = (tx / 16) - 6;
    int r2 = i * i + j * j;
    if (r2 < 36) {
      float gweight = exp(-r2 / (2.5f * 2.5f * 2.0f));
      int pos = (y + step * j) * pitch + (x + step * i);
      float dx = gweight * dxd[pos];
      float dy = gweight * dyd[pos];
      float angle = atan2(dy, dx);
      int a = max(min((int)(angle * (21 / CV_PI)) + 21, 41), 0);
      atomicAdd(resx + a, dx);
      atomicAdd(resy + a, dy);
    }
    __syncthreads();
    if (tx < 42) {
      re8x[tx] = resx[tx];
      re8y[tx] = resy[tx];
      for (int k = tx + 1; k < tx + 7; k++) {
        re8x[tx] += resx[k < 42 ? k : k - 42];
        re8y[tx] += resy[k < 42 ? k : k - 42];
      }
    }
    __syncthreads();
    if (tx == 0) {
      float maxr = 0.0f;
      int maxk = 0;
      for (int k = 0; k < 42; k++) {
        float r = re8x[k] * re8x[k] + re8y[k] * re8y[k];
        if (r > maxr) {
          maxr = r;
          maxk = k;
        }
      }
      float angle = atan2(re8y[maxk], re8x[maxk]);
      d_pts[p].angle = (angle < 0.0f ? angle + 2.0f * CV_PI : angle);
      // printf("XXX %.2f %.2f %.2f\n", d_pts[p].pt.x, d_pts[p].pt.y,
      // d_pts[p].angle/CV_PI*180.0f);
    }
  }

  double FindOrientation(std::vector<cv::KeyPoint> & h_pts,
                         cv::KeyPoint * d_pts, std::vector<CudaImage> & h_imgs,
                         CudaImage * d_imgs) {
    safeCall(hipMemcpyAsync(d_imgs, (float *)&h_imgs[0],
                             sizeof(CudaImage) * h_imgs.size(),
                             hipMemcpyHostToDevice));
    int numPts = h_pts.size();
    // TimerGPU timer0(0);
    dim3 blocks(numPts);
    dim3 threads(ORIENT_S);
    FindOrientation << <blocks, threads>>> (d_pts, d_imgs);
    // checkMsg("FindOrientation() execution failed\n");
    // safeCall(hipDeviceSynchronize());
    double gpuTime = 0;  // timer0.read();
#ifdef VERBOSE
    printf("FindOrientation time =        %.2f ms\n", gpuTime);
#endif
    return gpuTime;
  }
